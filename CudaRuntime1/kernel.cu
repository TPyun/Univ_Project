#include "hip/hip_runtime.h"
﻿//#include <iostream>
//#include <stdio.h>
//#include <random>
//#include <windows.h>
//#include <time.h>
//#include <hip/hip_runtime.h>
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
//using namespace std;
//
//const int one_side_number = 1000;
//const int max_height = 8;
//const int base_floor = 1;
//
//random_device rd;
// default_random_engine dre(rd());
//
//uniform_int_distribution <int>floor_uid(1, 2);
//
//uniform_int_distribution <int>hill_size_uid(6, 15);
//uniform_int_distribution <int>hills_location(0, one_side_number);
//uniform_int_distribution <int>number_of_hills_uid(5, 15);
// uniform_int_distribution <int>height_uid(5, max_height);
//
//char terrain_array_host[one_side_number * one_side_number]{};
//
//void show_array()
//{
//	for (int i = 0; i < one_side_number; ++i) {
//		for (int j = 0; j < one_side_number; ++j) {
//			printf("%d ", terrain_array_host[i * one_side_number + j]);
//		}
//		printf("\n");
//	}
//}
//
//void add_floor()
//{
//	for (int i = 0; i < one_side_number * one_side_number; i++){
//		terrain_array_host[i] += base_floor;
//	}
//}
//
//__global__ 
//void make_hills_cuda(int hill_location_x, int hill_location_y, int radius, int height, char* terrain_array_device_gpu)
//{
//	if ((int)terrain_array_device_gpu[hill_location_x * one_side_number + hill_location_y] + height > max_height) {
//		printf("%d, %d\n", terrain_array_device_gpu[hill_location_x * one_side_number + hill_location_y], height);
//		return;
//	}
//
//	for (int j = 0; j < one_side_number; j++) {
//		for (int k = 0; k < one_side_number; k++) {
//			int distance = sqrt(pow(j - hill_location_x, 2) + pow(k - hill_location_y, 2));
//			if (distance <= radius) {
//				terrain_array_device_gpu[j * one_side_number + k] += (height - 1) * (radius - distance) / radius + 1;
//				if ((int)terrain_array_device_gpu[j * one_side_number + k] > max_height) {
//					terrain_array_device_gpu[j * one_side_number + k] = (char)max_height;
//				}
//			}
//		}
//	}
//	/*for (int i = 0; i < one_side_number; ++i) {
//		for (int j = 0; j < one_side_number; ++j) {
//			printf("%d ", terrain_array_device_gpu[i * one_side_number + j]);
//		}
//		printf("\n");
//	}*/
//}
//
// void make_hills()
//{
//	char* terrain_array_device_gpu;
//	hipMalloc(&terrain_array_device_gpu, one_side_number * one_side_number * sizeof(char));
//	hipMemcpy(terrain_array_device_gpu, terrain_array_host, one_side_number * one_side_number * sizeof(char), hipMemcpyHostToDevice);
//	 
//	for (int i = 0; i < number_of_hills_uid(dre); i++){
//		int height = height_uid(dre);
//		int radius = hill_size_uid(dre);
//		
//		int hill_location_x = hills_location(dre);
//		int hill_location_y = hills_location(dre);
//
//		make_hills_cuda <<<1, 1>>> ( hill_location_x, hill_location_y, radius, height, terrain_array_device_gpu);
//	}
//	hipMemcpy(terrain_array_host, terrain_array_device_gpu, one_side_number * one_side_number * sizeof(char), hipMemcpyDeviceToHost);
//}
//
//int main()
//{	
//	clock_t terrain_generate_start_time = clock();
//	make_hills();
//	clock_t  terrain_generate_end_time = clock();
//	clock_t memcpy_end_time = clock();
//	//add_floor();
//
//	show_array();
//	cout << "Terrain Generation Complete" << endl;
//	cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
//	cout << "Terrain Array Size : " << sizeof(terrain_array_host) << " Bytes" << endl;
//	cout << "Terrain Generate Time : " << (double)(terrain_generate_end_time - terrain_generate_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
//	cout << "Memcpy Time : " << (double)(memcpy_end_time - terrain_generate_end_time) / CLOCKS_PER_SEC << " Seconds" << endl;
//}
//


//#include <iostream>
//#include <stdio.h>
//#include <random>
//#include <windows.h>
//#include <time.h>
//#include <hip/hip_runtime.h>
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
//using namespace std;
//
//const int one_side_number = 1000;
//const int max_height = 8;
//const int base_floor = 1;
//
//random_device rd;
//default_random_engine dre(rd());
//
//uniform_int_distribution <int>floor_uid(1, 2);
//
//uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
//uniform_int_distribution <int>hills_location(0, one_side_number);
//uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 5);
//uniform_int_distribution <int>height_uid(5, max_height);
//
////uniform_int_distribution <int>hill_size_uid(20, 50);
////uniform_int_distribution <int>hills_location(0, one_side_number);
////uniform_int_distribution <int>number_of_hills_uid(10, 20);
////uniform_int_distribution <int>height_uid(5, max_height);
//
//__device__ char terrain_array_device[one_side_number * one_side_number]{};
//char terrain_array_host[one_side_number * one_side_number]{};
//
//void show_array()
//{
//	for (int i = 0; i < one_side_number; ++i) {
//		for (int j = 0; j < one_side_number; ++j) {
//			printf("%d ", terrain_array_host[i * one_side_number + j]);
//		}
//		printf("\n");
//	}
//}
//
//void add_floor()
//{
//	for (int i = 0; i < one_side_number * one_side_number; i++) {
//		terrain_array_host[i] += base_floor;
//	}
//}
//
////__device__ int getRand(hiprandState* s, int A, int B)
////{
////	float rand_int = hiprand_uniform(s);
////	rand_int = rand_int * (B - A) + A;
////	return rand_int;
////}
//
//__global__ void make_hills_cuda(int hill_location_x, int hill_location_y, int radius, int height, int i)
//{
//	/*int id = threadIdx.x + blockDim.x * blockDim.y;
//	unsigned int seed = id;
//	hiprandState s;
//	hiprand_init(seed, 0, 0, &s);
//	int rand_num = getRand(&s, 0, 10);
//	printf("%d\n", rand_num);*/
//
//	printf("%d 번째 언덕 생성 시작\n", i);
//	if (terrain_array_device[hill_location_x * one_side_number + hill_location_y] + height > max_height) {
//		printf("%d, %d\n", terrain_array_device[hill_location_x * one_side_number + hill_location_y], height);
//		return;
//	}
//
//	//for문 말고 멀티쓰레딩으로 하자
//	for (int j = 0; j < one_side_number; j++) {
//		for (int k = 0; k < one_side_number; k++) {
//			int distance = sqrt(pow(j - hill_location_x, 2) + pow(k - hill_location_y, 2));
//			if (distance <= radius) {
//				terrain_array_device[j * one_side_number + k] += (height - 1) * (radius - distance) / radius + 1;
//				if (terrain_array_device[j * one_side_number + k] > max_height) {
//					terrain_array_device[j * one_side_number + k] = max_height;
//				}
//			}
//		}
//	}
//}
//
//void make_hills()
//{
//	for (int i = 0; i < number_of_hills_uid(dre); i++) {
//		int height = height_uid(dre);
//		int radius = hill_size_uid(dre);
//
//		int hill_location_x = hills_location(dre);
//		int hill_location_y = hills_location(dre);
//
//		make_hills_cuda << <1, 1 >> > (hill_location_x, hill_location_y, radius, height, i);
//	}
//}
//
//int main()
//{
//	clock_t terrain_generate_start_time = clock();
//
//	make_hills();
//	clock_t  terrain_generate_end_time = clock();
//	cout << "Terrain Generation Complete" << endl;
//
//	hipMemcpyFromSymbol(&terrain_array_host, HIP_SYMBOL(terrain_array_device), one_side_number * one_side_number * sizeof(char), 0, hipMemcpyDeviceToHost);
//	clock_t memcpy_end_time = clock();
//	printf("Memcpy Complete\n");
//
//	//add_floor();
//
//	//show_array();
//	cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
//	cout << "Terrain Array Size : " << sizeof(terrain_array_host) << " Bytes" << endl;
//	cout << "Terrain Generate Time : " << (double)(terrain_generate_end_time - terrain_generate_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
//	cout << "Memcpy Time : " << (double)(memcpy_end_time - terrain_generate_end_time) / CLOCKS_PER_SEC << " Seconds" << endl;
//}


//#include <iostream>
//#include <stdio.h>
//#include <random>
//#include <windows.h>
//#include <time.h>
//#include <hip/hip_runtime.h>
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
//#include <hip/hip_cooperative_groups.h>
//#include ""
//using namespace std;
//
//const unsigned int one_side_number = 4000;
//
//const int max_height = 9;
//const int base_floor = 1;
//
//random_device rd;
//default_random_engine dre(rd());
//
//uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 5);
//
//char terrain_array_host[one_side_number * one_side_number]{};
//__device__ char terrain_array_device[one_side_number * one_side_number];
//
//void get_device_info()
//{
//	hipDeviceProp_t  prop;
//
//	int count;
//	hipGetDeviceCount(&count);
//
//	for (int i = 0; i < count; i++) {
//		hipGetDeviceProperties(&prop, i);
//		printf("   --- General Information for device %d ---\n", i);
//		printf("Name:  %s\n", prop.name);
//		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
//		printf("Clock rate:  %d\n", prop.clockRate);
//		printf("Device copy overlap:  ");
//		if (prop.deviceOverlap)
//			printf("Enabled\n");
//		else
//			printf("Disabled\n");
//		printf("Kernel execution timeout :  ");
//		if (prop.kernelExecTimeoutEnabled)
//			printf("Enabled\n");
//		else
//			printf("Disabled\n");
//		printf("\n");
//
//		printf("   --- Memory Information for device %d ---\n", i);
//		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
//		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
//		printf("Max mem pitch:  %ld\n", prop.memPitch);
//		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
//		printf("\n");
//
//		printf("   --- MP Information for device %d ---\n", i);
//		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
//		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
//		printf("Registers per mp:  %d\n", prop.regsPerBlock);
//		printf("Threads in warp:  %d\n", prop.warpSize);
//		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
//		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
//		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
//		printf("\n");
//	}
//}
//
//void show_array(char* terrain_array_host)
//{
//	for (int i = 0; i < one_side_number; ++i) {
//		for (int j = 0; j < one_side_number; ++j) {
//			printf("%d ", terrain_array_host[i * one_side_number + j]);
//		}
//		printf("\n");
//	}
//}
//
//__device__ 
//int getRand(hiprandState* s, int A, int B)
//{
//	float rand_int = hiprand_uniform(s);
//	rand_int = rand_int * (B+1 - A) + A;
//	return rand_int;
//}
//
//__global__
//void add_floor(char* terrain_array_host)
//{
//	for (int i = 0; i < one_side_number * one_side_number; i++) {
//		terrain_array_host[i] += base_floor;
//	}
//}
//
//__global__
//void make_hills_cuda()
//{
//	int id = threadIdx.x + blockIdx.x * blockDim.x;
//	hiprandState s;
//	hiprand_init(id, 0, 0, &s);
//
//	/*int hill_location_x = 500;
//	int hill_location_y = 500;
//	int radius = getRand(&s, one_side_number / 20, one_side_number / 10);
//	int height = getRand(&s, 5, max_height);*/
//
//	int hill_location_x = getRand(&s, 0, one_side_number);
//	int hill_location_y = getRand(&s, 0, one_side_number);
//	int radius = getRand(&s, one_side_number / 20, one_side_number / 10);
//	int height = getRand(&s, 5, max_height);
//
//	//printf("START thread: %3d seed: %3d x: %3d y: %3d radius: %3d height: %3d\n", threadIdx.x + blockIdx.x * blockDim.x,id, hill_location_x, hill_location_y, radius, height);
//	printf("%d", terrain_array_device[hill_location_x * one_side_number + hill_location_y]);
//
//	if (terrain_array_device[hill_location_x * one_side_number + hill_location_y]>0) {
//		printf("%d", terrain_array_device[hill_location_x * one_side_number + hill_location_y]);
//		printf("fail\n");
//	}
//	else {
//		for (int j = 0; j < one_side_number; j++) {
//			for (int k = 0; k < one_side_number; k++) {
//				int distance = sqrt(pow(j - hill_location_x, 2) + pow(k - hill_location_y, 2));
//				if (distance <= radius) {
//					terrain_array_device[j * one_side_number + k] += (height - 1) * (radius - distance) / radius + 1;
//					/*if (terrain_array_device[j * one_side_number + k] > max_height) {
//						terrain_array_device[j * one_side_number + k] = max_height;
//					}*/
//				}
//			}
//		}
//	}
//}
//
//
//int main()
//{
//	//get_device_info();
//	/*char* terrain_array_host;
//	hipHostMalloc((void**)&terrain_array_host, one_side_number * one_side_number * sizeof(char));*/
//	//char* terrain_array_host = new char[one_side_number * one_side_number];
//	
//	clock_t terrain_generate_start_time = clock();
//	make_hills_cuda << <60, 5 >> > ();
//
//	hipMemcpyFromSymbol(&terrain_array_host, HIP_SYMBOL(terrain_array_device), one_side_number * one_side_number * sizeof(char), 0, hipMemcpyDeviceToHost);
//	printf("Terrain Generation Complete\n");
//
//	clock_t  terrain_generate_end_time = clock();
//
//	//add_floor << <1, 1 >> > (terrain_array_host);
//
//	//show_array(terrain_array_host);
//	cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
//	cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
//	cout << "Terrain Generate Time : " << (double)(terrain_generate_end_time - terrain_generate_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
//
//}

//#include <iostream>
//#include <stdio.h>
//#include <random>
//#include <windows.h>
//#include <time.h>
//#include <hip/hip_runtime.h>
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
//#include <hip/hip_cooperative_groups.h>
//#include ""
//using namespace std;
//
//const unsigned int one_side_number = 1000;
//
//const int max_height = 9;
//const int base_floor = 1;
//
//typedef struct hill_info{
//	int x;
//	int y;
//	int radius;
//	int height;
//} HI;
//
//random_device rd;
//default_random_engine dre(rd());
//uniform_int_distribution <int>hills_location(0, one_side_number);
//uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 5);
//uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
//uniform_int_distribution <int>height_uid(5, max_height);
//
//char terrain_array_host[one_side_number * one_side_number]{};
//__device__ char terrain_array_device[one_side_number * one_side_number];
//HI hill_location_host[one_side_number * one_side_number];
//__device__ HI hill_location_device[one_side_number * one_side_number];
//
//void get_device_info()
//{
//	hipDeviceProp_t  prop;
//
//	int count;
//	hipGetDeviceCount(&count);
//
//	for (int i = 0; i < count; i++) {
//		hipGetDeviceProperties(&prop, i);
//		printf("   --- General Information for device %d ---\n", i);
//		printf("Name:  %s\n", prop.name);
//		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
//		printf("Clock rate:  %d\n", prop.clockRate);
//		printf("Device copy overlap:  ");
//		if (prop.deviceOverlap)
//			printf("Enabled\n");
//		else
//			printf("Disabled\n");
//		printf("Kernel execution timeout :  ");
//		if (prop.kernelExecTimeoutEnabled)
//			printf("Enabled\n");
//		else
//			printf("Disabled\n");
//		printf("\n");
//
//		printf("   --- Memory Information for device %d ---\n", i);
//		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
//		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
//		printf("Max mem pitch:  %ld\n", prop.memPitch);
//		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
//		printf("\n");
//
//		printf("   --- MP Information for device %d ---\n", i);
//		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
//		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
//		printf("Registers per mp:  %d\n", prop.regsPerBlock);
//		printf("Threads in warp:  %d\n", prop.warpSize);
//		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
//		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
//		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
//		printf("\n");
//	}
//}
//
//void show_array(char* terrain_array_host)
//{
//	for (int i = 0; i < one_side_number; ++i) {
//		for (int j = 0; j < one_side_number; ++j) {
//			printf("%d ", terrain_array_host[i * one_side_number + j]);
//		}
//		printf("\n");
//	}
//}
//
//__device__
//int getRand(hiprandState* s, int A, int B)
//{
//	float rand_int = hiprand_uniform(s);
//	rand_int = rand_int * (B + 1 - A) + A;
//	return rand_int;
//}
//
//__global__
//void add_floor()
//{
//	for (int i = 0; i < one_side_number * one_side_number; i++) {
//		terrain_array_device[i] += base_floor;
//	}
//}
//
//__global__
//void make_hills_cuda()
//{	
//	int id = threadIdx.x + blockIdx.x * blockDim.x;
//	hiprandState s;
//
//	int hill_location_x = hill_location_device[id].x;
//	int hill_location_y = hill_location_device[id].y;
//	int radius = hill_location_device[id].radius;
//	int height = hill_location_device[id].height;
//
//	printf("START thread: %3d seed: %3d x: %3d y: %3d radius: %3d height: %3d\n", threadIdx.x + blockIdx.x * blockDim.x,id, hill_location_x, hill_location_y, radius, height);
//	
//	for (int j = 0; j < one_side_number; j++) {
//		for (int k = 0; k < one_side_number; k++) {
//			int distance = sqrt(pow(j - hill_location_x, 2) + pow(k - hill_location_y, 2));
//			if (distance <= radius) {
//				terrain_array_device[j * one_side_number + k] += (height - 1) * (radius - distance) / radius + 1;
//				if (terrain_array_device[j * one_side_number + k] > max_height) {
//					terrain_array_device[j * one_side_number + k] = max_height;
//				}
//			}
//		}
//	}
//}
//
//int make_hill_location()
//{
//	int num_of_hills = number_of_hills_uid(dre);
//
//	cout << "expected num of hill: " << num_of_hills << endl;
//
//	for (int i = 0; i < num_of_hills; i++) {
//		hill_location_host[i].x = hills_location(dre);
//		hill_location_host[i].y = hills_location(dre);
//		hill_location_host[i].radius = hill_size_uid(dre);
//		hill_location_host[i].height = height_uid(dre);
//	}
//	for (int i = 0; i < num_of_hills; i++) {
//		sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
//	}
//	/*for (int i = 0; i < num_of_hills; i++) {
//		cout << hill_location_host[i].x << " " << hill_location_host[i].y << endl;
//	}*/
//	for (int a = 0; a < num_of_hills; a++) {
//		for (int b = 0; b < num_of_hills; b++) {
//			/*cout << pow(hill_location_host[j].x - hill_location_host[k].x, 2) << " " << pow(hill_location_host[j].radius + hill_location_host[k].radius, 2) << endl;
//			cout << pow(hill_location_host[j].y - hill_location_host[k].y, 2) << " " << pow(hill_location_host[j].radius + hill_location_host[k].radius, 2) << endl;
//			cout << endl;*/
//			if (a != b) {
//				if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
//					if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
//						//cout << "hit" << endl;
//						for (int i = b; i < num_of_hills; i++) {
//							hill_location_host[i] = hill_location_host[i + 1];
//						}
//						b -= 1;
//						num_of_hills -= 1;
//					}
//				}
//			}
//		}
//	}
//	hipMemcpyToSymbol(HIP_SYMBOL(hill_location_device), &hill_location_host, num_of_hills * sizeof(HI), 0, hipMemcpyHostToDevice);
//	cout << "real num of hill: " << num_of_hills << endl;
//	return num_of_hills;
//}
//
//int main()
//{
//	//get_device_info();
//
//	int num_of_hills = make_hill_location();
//
//	clock_t terrain_generate_start_time = clock();
//	make_hills_cuda << <2, num_of_hills / 2 >> > ();
//
//	hipMemcpyFromSymbol(&terrain_array_host, HIP_SYMBOL(terrain_array_device), one_side_number * one_side_number * sizeof(char), 0, hipMemcpyDeviceToHost);
//	printf("Terrain Generation Complete\n");
//
//	clock_t  terrain_generate_end_time = clock();
//
//	//add_floor << <1, 1 >> > (terrain_array_host);
//
//	//show_array(terrain_array_host);
//	cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
//	cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
//	cout << "Terrain Generate Time : " << (double)(terrain_generate_end_time - terrain_generate_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
//
//}

//#include <iostream>
//#include <stdio.h>
//#include <random>
//#include <windows.h>
//#include <time.h>
//#include <hip/hip_runtime.h>
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
//#include <hip/hip_cooperative_groups.h>
//#include ""
//using namespace std;
//
//const unsigned int one_side_number = 5000;
//
//const int max_height = 9;
//const int base_floor = 1;
//
//typedef struct hill_info {
//	int x;
//	int y;
//	int radius;
//	int height;
//} HI;
//
//random_device rd;
//default_random_engine dre(rd());
//uniform_int_distribution <int>hills_location(0, one_side_number);
//uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 5);
//uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
//uniform_int_distribution <int>height_uid(5, max_height);
//
//char terrain_array_host[one_side_number * one_side_number]{};
//__device__ char terrain_array_device[one_side_number * one_side_number];
//HI hill_location_host[one_side_number * one_side_number];
//__device__ HI hill_location_device[one_side_number * one_side_number];
//
//void get_device_info()
//{
//	hipDeviceProp_t  prop;
//
//	int count;
//	hipGetDeviceCount(&count);
//
//	for (int i = 0; i < count; i++) {
//		hipGetDeviceProperties(&prop, i);
//		printf("   --- General Information for device %d ---\n", i);
//		printf("Name:  %s\n", prop.name);
//		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
//		printf("Clock rate:  %d\n", prop.clockRate);
//		printf("Device copy overlap:  ");
//		if (prop.deviceOverlap)
//			printf("Enabled\n");
//		else
//			printf("Disabled\n");
//		printf("Kernel execution timeout :  ");
//		if (prop.kernelExecTimeoutEnabled)
//			printf("Enabled\n");
//		else
//			printf("Disabled\n");
//		printf("\n");
//
//		printf("   --- Memory Information for device %d ---\n", i);
//		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
//		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
//		printf("Max mem pitch:  %ld\n", prop.memPitch);
//		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
//		printf("\n");
//
//		printf("   --- MP Information for device %d ---\n", i);
//		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
//		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
//		printf("Registers per mp:  %d\n", prop.regsPerBlock);
//		printf("Threads in warp:  %d\n", prop.warpSize);
//		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
//		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
//		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
//		printf("\n");
//	}
//}
//
//void show_array(char* terrain_array_host)
//{
//	for (int i = 0; i < one_side_number; ++i) {
//		for (int j = 0; j < one_side_number; ++j) {
//			printf("%d ", terrain_array_host[i * one_side_number + j]);
//		}
//		printf("\n");
//	}
//}
//
//__device__
//int getRand(hiprandState* s, int A, int B)
//{
//	float rand_int = hiprand_uniform(s);
//	rand_int = rand_int * (B + 1 - A) + A;
//	return rand_int;
//}
//
//__global__
//void add_floor()
//{
//	for (int i = 0; i < one_side_number * one_side_number; i++) {
//		terrain_array_device[i] += base_floor;
//	}
//}
//
//__global__
//void make_hills_cuda()
//{
//	int id = threadIdx.x + blockIdx.x * blockDim.x;
//	hiprandState s;
//
//	int hill_location_x = hill_location_device[id].x;
//	int hill_location_y = hill_location_device[id].y;
//	int radius = hill_location_device[id].radius;
//	int height = hill_location_device[id].height;
//	int distance{};
//
//	printf("START thread: %3d seed: %3d x: %3d y: %3d radius: %3d height: %3d\n", threadIdx.x + blockIdx.x * blockDim.x, id, hill_location_x, hill_location_y, radius, height);
//
//	for (int x = hill_location_x - radius; x <= hill_location_x + radius; x++) {
//		for (int y = hill_location_y - radius ; y <= hill_location_y + radius; y++) {
//			distance = sqrt(pow(x - hill_location_x, 2) + pow(y - hill_location_y, 2));
//			if (distance <= radius) {
//				//printf("%d %d\n", x, y);
//				terrain_array_device[x * one_side_number + y] += (height - 1) * (radius - distance) / radius + 1;
//				//printf("%d\n",(height - 1) * (radius - distance) / radius + 1);
//				if (terrain_array_device[x * one_side_number + y] > max_height) {
//					terrain_array_device[x * one_side_number + y] = max_height;
//				}
//			}
//		}
//	}
//}
//
//int make_hill_location()
//{
//	int num_of_hills = number_of_hills_uid(dre);
//
//	cout << "expected num of hill: " << num_of_hills << endl;
//
//	for (int i = 0; i < num_of_hills; i++) {
//		hill_location_host[i].x = hills_location(dre);
//		hill_location_host[i].y = hills_location(dre);
//		hill_location_host[i].radius = hill_size_uid(dre);
//		hill_location_host[i].height = height_uid(dre);
//	}
//	for (int i = 0; i < num_of_hills; i++) {
//		sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
//	}
//
//	for (int a = 0; a < num_of_hills; a++) {
//		for (int b = 0; b < num_of_hills; b++) {
//			if (a != b) {
//				if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
//					if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
//						//cout << "hit" << endl;
//						for (int i = b; i < num_of_hills; i++) {
//							hill_location_host[i] = hill_location_host[i + 1];
//						}
//						b -= 1;
//						num_of_hills -= 1;
//					}
//				}
//			}
//		}
//	}
//	for (int i = 0; i < num_of_hills; i++) {
//		if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
//			for (int j = i; j < num_of_hills; j++) {
//				hill_location_host[j] = hill_location_host[j + 1];
//			}
//			i -= 1;
//			num_of_hills -= 1;
//		}
//		else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
//			for (int j = i; j < num_of_hills; j++) {
//				hill_location_host[j] = hill_location_host[j + 1];
//			}
//			i -= 1;
//			num_of_hills -= 1;
//		}
//		else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
//			for (int j = i; j < num_of_hills; j++) {
//				hill_location_host[j] = hill_location_host[j + 1];
//			}
//			i -= 1;
//			num_of_hills -= 1;
//		}
//		else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
//			for (int j = i; j < num_of_hills; j++) {
//				hill_location_host[j] = hill_location_host[j + 1];
//			}
//			i -= 1;
//			num_of_hills -= 1;
//		}
//	}
//	hipMemcpyToSymbol(HIP_SYMBOL(hill_location_device), &hill_location_host, num_of_hills * sizeof(HI), 0, hipMemcpyHostToDevice);
//	cout << "real num of hill: " << num_of_hills << endl;
//	return num_of_hills;
//}
//
//int main()
//{
//	//get_device_info();
//
//	int num_of_hills = make_hill_location();
//
//	clock_t terrain_generate_start_time = clock();
//	make_hills_cuda << <num_of_hills/2, 2>> > ();
//	//add_floor << <1, 1 >> > ();
//
//	hipMemcpyFromSymbol(&terrain_array_host, HIP_SYMBOL(terrain_array_device), one_side_number * one_side_number * sizeof(char), 0, hipMemcpyDeviceToHost);
//	printf("Terrain Generation Complete\n");
//
//	clock_t  terrain_generate_end_time = clock();
//
//	//show_array(terrain_array_host);
//	
//	cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
//	cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
//	cout << "Terrain Generate Time : " << (double)(terrain_generate_end_time - terrain_generate_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
//}


//2차원 배열
#include <iostream>
#include <stdio.h>
#include <random>
#include <windows.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include ""
using namespace std;

const unsigned int one_side_number = 1000;

const int max_height = 9;
const int base_floor = 1;

typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>hills_location(0, one_side_number);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 5);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(5, max_height);

char terrain_array_host[one_side_number][one_side_number]{};
__device__ char terrain_array_device[one_side_number][one_side_number];

HI hill_location_host[one_side_number * one_side_number];
__device__ HI hill_location_device[one_side_number * one_side_number];

void get_device_info()
{
	hipDeviceProp_t  prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
		printf("\n");

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}

void show_array()
{
	for (int x = 0; x < one_side_number; x++) {
		for (int y = 0; y < one_side_number; y++) {
			printf("%d ", terrain_array_host[x][y]);
		}
		printf("\n");
	}
}

__global__
void add_floor()
{
	for (int x = 0; x < one_side_number; x++) {
		for (int y = 0; y < one_side_number; y++) {
			terrain_array_device[x][y] += base_floor;
		}
	}
}

__global__
void make_hills_cuda()
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState s;

	int hill_location_x = hill_location_device[id].x;
	int hill_location_y = hill_location_device[id].y;
	int radius = hill_location_device[id].radius;
	int height = hill_location_device[id].height;
	int distance{};

	printf("START thread: %3d seed: %3d x: %3d y: %3d radius: %3d height: %3d\n", threadIdx.x + blockIdx.x * blockDim.x, id, hill_location_x, hill_location_y, radius, height);

	for (int x = hill_location_x - radius; x <= hill_location_x + radius; x++) {
		for (int y = hill_location_y - radius; y <= hill_location_y + radius; y++) {
			distance = sqrt(pow(x - hill_location_x, 2) + pow(y - hill_location_y, 2));
			if (distance <= radius) {
				//printf("%d %d\n", x, y);
				terrain_array_device[x][y] += (height - 1) * (radius - distance) / radius + 1;
				//printf("%d\n",(height - 1) * (radius - distance) / radius + 1);
				if (terrain_array_device[x][y] > max_height) {
					terrain_array_device[x][y] = max_height;
				}
			}
		}
	}
}

int make_hill_location()
{
	int num_of_hills = number_of_hills_uid(dre);

	cout << "expected num of hill: " << num_of_hills << endl;

	for (int i = 0; i < num_of_hills; i++) {
		hill_location_host[i].x = hills_location(dre);
		hill_location_host[i].y = hills_location(dre);
		hill_location_host[i].radius = hill_size_uid(dre);
		hill_location_host[i].height = height_uid(dre);
	}
	for (int i = 0; i < num_of_hills; i++) {
		sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
	}
	for (int a = 0; a < num_of_hills; a++) {
		for (int b = 0; b < num_of_hills; b++) {
			if (a != b) {
				if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
					if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						//cout << "hit" << endl;
						for (int i = b; i < num_of_hills; i++) {
							hill_location_host[i] = hill_location_host[i + 1];
						}
						b -= 1;
						num_of_hills -= 1;
					}
				}
			}
		}
	}
	for (int i = 0; i < num_of_hills; i++) {
		if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(hill_location_device), &hill_location_host, num_of_hills * sizeof(HI), 0, hipMemcpyHostToDevice);
	cout << "real num of hill: " << num_of_hills << endl;
	return num_of_hills;
}

int main()
{
	//get_device_info();

	int num_of_hills = make_hill_location();

	clock_t terrain_generate_start_time = clock();
	make_hills_cuda << <num_of_hills / 2, 2 >> > ();
	//add_floor << <1, 1 >> > ();

	hipMemcpyFromSymbol(&terrain_array_host, HIP_SYMBOL(terrain_array_device), one_side_number * one_side_number * sizeof(char), 0, hipMemcpyDeviceToHost);
	printf("Terrain Generation Complete\n");

	clock_t  terrain_generate_end_time = clock();

	show_array();

	cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
	cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
	cout << "Terrain Generate Time : " << (double)(terrain_generate_end_time - terrain_generate_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
}
