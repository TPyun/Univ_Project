#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <random>
#include <windows.h>
#include <time.h>
#include <math.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include ""
#define PI 3.1415926
using namespace std;

const int one_side_number = 40000;	//39936
const int player_sight_size = 1000;	//1024 넘으면 안됨

const int max_height = 8;
const int base_floor = 1;

typedef struct two_int {
	int x;
	int y;
} TI;

typedef struct two_float {
	float x;
	float y;
} TF;

typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>hills_location(0, one_side_number);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 10);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(5, max_height);

uniform_int_distribution <int>wind_speed_uid(0, 50);
uniform_int_distribution <int>wind_angle_uid(0, 360);


void get_device_info()
{
	hipDeviceProp_t  prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
		printf("\n");

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}

void show_array(char** terrain_array_host, int size)
{
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			printf("%d ", terrain_array_host[x][y]);
		}
		printf("\n");
	}
}

__global__
void player_terrain_update_cuda(char** terrain_player_sight_device, HI* hill_location_device, int num_of_hills, TI player_location, TF wind_direction, int wind_speed)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int terrain_x = player_location.x + x;
	int terrain_y = player_location.y + y;
	
	if (terrain_x >= 0 && terrain_x <= one_side_number && terrain_y >= 0 && terrain_y <= one_side_number) {
		terrain_player_sight_device[x][y] = base_floor;
		for (int i = 0; i < num_of_hills; i++) {
			
			//원래 언덕 채우기
			int hill_location_x = hill_location_device[i].x;
			int hill_location_y = hill_location_device[i].y;
			int radius = hill_location_device[i].radius;
			int height = hill_location_device[i].height;
			int distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

			if (distance <= radius) {
				terrain_player_sight_device[x][y] += (height) * (radius - distance) / radius;

				//언덕 깎기
				hill_location_x = hill_location_device[i].x - radius * wind_direction.x * (100 - wind_speed) / 50;
				hill_location_y = hill_location_device[i].y - radius * wind_direction.y * (100 - wind_speed) / 50;
				distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

				if (distance <= radius) {
					terrain_player_sight_device[x][y] -= height * (radius - distance) / radius + base_floor;
					if (terrain_player_sight_device[x][y] > max_height) {
						terrain_player_sight_device[x][y] = max_height;
					}
					else if (terrain_player_sight_device[x][y] < base_floor) {
						terrain_player_sight_device[x][y] = base_floor;
					}
				}
			}
		}
	}
	else {
		//맵 밖의 경우 0
		terrain_player_sight_device[x][y] = 0;
	}
}

template <typename T>
int delete_array(T* array, int i, int size)
{
	for (int j = i; j < size; j++) {
		array[j] = array[j + 1];
	}
	size -= 1;
	return size;
}

int make_hill_location(HI* hill_location_host)
{
	int num_of_hills = number_of_hills_uid(dre);
	cout << "expected num of hills: " << num_of_hills << endl;

	for (int i = 0; i < num_of_hills; i++) {
		hill_location_host[i].x = hills_location(dre);
		hill_location_host[i].y = hills_location(dre);
		hill_location_host[i].radius = hill_size_uid(dre);
		hill_location_host[i].height = height_uid(dre);
	}
	//sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
	for (int a = 0; a < num_of_hills; a++) {
		for (int b = 0; b < num_of_hills; b++) {
			if (a != b) {
				if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
					if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						num_of_hills = delete_array(hill_location_host,b, num_of_hills);
						b--;
					}
				}
			}
		}
	}
	//for (int i = 0; i < num_of_hills; i++) {
	//	if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
	//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//	else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
	//		num_of_hills = delete_array (hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//	else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
	//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//	else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
	//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//}
	cout << "real num of hills: " << num_of_hills << endl;
	return num_of_hills;
}

void make_new_hills(HI* hill_location_host, int& num_of_hills, int origin_num_of_hills, TF wind_direction, int wind_speed)
{
	cout << "Wind info: " << wind_direction.x << " " << wind_direction.y << endl;
	if (wind_speed) {
		hill_location_host[num_of_hills].radius = hill_size_uid(dre);
		hill_location_host[num_of_hills].height = height_uid(dre);

		hill_location_host[num_of_hills].x = hills_location(dre);
		hill_location_host[num_of_hills].y = hills_location(dre);
		
		while (1) {
			hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
			hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
			if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
				break;
			}
			 if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
				break;
			}
			if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
				break;
			}
			if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
				break;
			}
		}
		
		int collide_iter{};
		for (int a = 0; a < num_of_hills; a++) {
			if (collide_iter > 10) {	//무한루프 빠질 가능성으로 인해 횟수 제한
				return;
			}
			//cout << "대상: "<<a << " " << hill_location_host[a].x << " " << hill_location_host[a].y << " " << hill_location_host[a].radius << " " << hill_location_host[a].height << endl;
			//cout << " 나: "<< num_of_hills<< " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills ].y << " " << hill_location_host[num_of_hills ].radius << " " << hill_location_host[num_of_hills].height << endl << endl;

			if (pow(hill_location_host[a].x - hill_location_host[num_of_hills].x, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {
				if (pow(hill_location_host[a].y - hill_location_host[num_of_hills].y, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {
					
					hill_location_host[num_of_hills].radius = hill_size_uid(dre);
					hill_location_host[num_of_hills].height = height_uid(dre);
					
					hill_location_host[num_of_hills ].x = hills_location(dre);
					hill_location_host[num_of_hills].y = hills_location(dre);
					
					while (1) {
						hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
						hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
						if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
							break;
						}
						if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
							break;
						}
						if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
							break;
						}
						if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
							break;
						}
					}
					cout << "충돌로 인해 바꿈" << endl;
					a = -1;
					collide_iter++;
				}
			}

		}
		cout << " 최종: " << num_of_hills << " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills].y << " " << hill_location_host[num_of_hills].radius << " " << hill_location_host[num_of_hills].height << endl << endl;
		num_of_hills++;
	}
}

void move_terrain(HI* hill_location_host, int& num_of_hills, TF wind_direction, int wind_speed)
{
	if (wind_speed) {
		int wind_move_x = wind_speed * wind_direction.x;
		int wind_move_y = wind_speed * wind_direction.y;
		cout<< "Wind == " << "X: " << wind_move_x << " " << "Y: " << wind_move_y << endl;
		for (int i = 0; i < num_of_hills; i++) {
			hill_location_host[i].x += wind_move_x;
			hill_location_host[i].y += wind_move_y;

			if ( hill_location_host[i].x - hill_location_host[i].radius > one_side_number) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			if (hill_location_host[i].x + hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			if ( hill_location_host[i].y - hill_location_host[i].radius > one_side_number) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			if (hill_location_host[i].y + hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
		}
		/*for (int i = 0; i < num_of_hills; i++) {
			cout << hill_location_host[i].x << " " << hill_location_host[i].y << endl;
		}
		cout << endl;*/
	}
}

void wind_decide(int& wind_speed, int& wind_angle)
{
	//wind speed 0-50
	//wind angle 0-360
	wind_speed = 50;// wind_speed_uid(dre);
	//wind_angle = 90; // wind_angle_uid(dre);
	wind_angle += 10;
	cout << wind_speed << " " << wind_angle << endl;
	//풍향을 언제마다 한번 업데이트 할 것인지, 풍속은 언제마다 한번 업데이트 할 것인지 회의를 통해 결정하자
	
	
}

int main()
{
	//get_device_info();

	
	//ToDo
	//마을 위치 랜덤 생성, 마을 위치는 높이 0인데 주변 언덕이 점차적인 높이로함, 언덕 움직이기
	// 언덕을 움직이는데 효울적인 방법: 플레이어가 보는 시야만 업데이트를 하게끔 해야함 
	// hill_location 업데이트는 항상 하고, cuda로 플레이어가 보는 시점만 terrain_array를 업데이트 해야함
	// 플레이어가 보는 시점을 1000*1000이라 가정하고, 그 부분의 terrain_array를 업데이트 해야함
	// 플레이어 수에 맞게 플레이어 시점 2차원 배열 terrain_array_for_player를 동적할당하여 생성해야함
	// 
	// 플레이어에게 맵 정보를 보내줘야 할 시기: 게임 시작, 지형이 변경될 때, 카메라가 움직일 때
	// 문제: 지형이 움직일때 terrain_array전체를 수정하고, 플레이어에게 terrain_array에 복사만 해서 일부분을 보여줄지,
	// hill_location을 수정하고, 플레이어에게 보여줄때, terrain_array를 아예 그릴지. ====이게 나을듯...40000*40000 만드는건 뻘짓이였다..
	//
	// 지형이 움직이면, 움직이는 방향의 반대 부분은 비게 된다. 언덕이 없다던가 등등...
	// 지형이 한 방향으로 움직이면 생성해줘야함
	// 그리고 40000 * 40000 밖으로 나간 언덕은  hill_location 배열에서 지워줘야함.
	// 
	// 언덕이 생성될때 언덕의 가파른 정도를 생각해보는 것도 좋을듯 하다
	// 
	// 사막의 모래언덕 모양을 위에서 보게 된다면 상현달~초승달의 모양을 띄게 되는것같다
	// 바람의 세기와 방향에 따라서 달 모양이 향하는 방향을 바꿔줘야한다.
	// 바람이 한번도 불지 않았다면 원 모양을 띈다는 전제하에 바람이 불면, 바람이 온 방향쪽을 깎아주는 모양을 만들자
	// 바람이 서쪽에서 분다면 언덕의 왼쪽 부분을 깎고 언덕이 오른쪽으로 이동하게끔 한다,
	// 그 후 바람이 동쪽에서 분다면 한칸씩 왼쪽은 차게되고, 오른쪽은 깎이는 형식으로 만들자.
	// 바람의 방향이 끼치는 영향: 언덕이 깎였던 부분이 채워지는 방향, 언덕이 깎이는 방향
	// 바람의 세기가 끼치는 영향: 지형이 이동하는 속도, 언덕이 깎이는 속도
	// 
	// //현재 언덕이 깎이는 모습을 제대로 보이게 하려면 풍속과 풍향을 확확 바꾸면 안된다.
	// 최대 풍속이 50이면 바람이 안부는 상태에서 10, 20, 30, 40, 50 이렇게 점점 늘려서 클라한테 보내줘야 한다
	// 아니면 언덕의 모양이 한번에 확 변한다
	// 
	// 블럭의 가로 세로 높이 비율을 1:1:0.2로 하면 어떨까 생각이 듦.
	// 만약 1:1:1이라면 붙어있는 블럭의 경사는 0, 45도 밖에 없음
	// 
	// 지형이 바람에 의해 이동하면, 이동한 방향의 반대 방향에 언덕이 새로 생성되어야 한다.
	// 처음에 생성된 언덕의 개수를 기억하고, 언덕이 사라지게 되면 그만큼 새로 생성되게끔 하자.
	// 왼쪽으로 바람이 불어 움직이고 있다면 오른쪽에서 생성, 아래로 바람이 분다면 위에서 생성 하게끔.
	// 언덕이 붙어서 생성되지 않게끔 해야한다. 추가를 할때 hill_location 배열을 확인해서 충돌검사를 하자.
	// 
	// 바람이 부는 알고리즘을 생성하자, 풍속과 풍향은 점차적으로 바뀌어야한다.
	// 
	// 
	// 
	//
	

	//Make Random Hills Information===================================================
	HI* hill_location_host = new HI[4000];
	HI* hill_location_device;
	hipMalloc((void**)&hill_location_device, 4000 * sizeof(HI));
	int num_of_hills = make_hill_location(hill_location_host);
	int origin_num_of_hills = num_of_hills;
	hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device
	printf("Random Hill Info Complete\n");

	
	//Terrain Memory Assignment For Player's Sight===================================================
	char** terrain_player_sight_host = new char* [player_sight_size];	// 2D array for host
	for (int i = 0; i < player_sight_size; i++) {
		terrain_player_sight_host[i] = new char[player_sight_size];
	}
	for (int i = 0; i < player_sight_size; i++) {
		for (int j = 0; j < player_sight_size; j++) {
			terrain_player_sight_host[i][j] = 0;
		}
	}
	char** terrain_player_sight_device;						// 2D array for device
	char* terrain_player_sight_temp[player_sight_size];		// 1D array temp
	hipMalloc((void**)&terrain_player_sight_device, player_sight_size * sizeof(char*));
	for (int i = 0; i < player_sight_size; i++) {
		hipMalloc((void**)&terrain_player_sight_temp[i], player_sight_size * sizeof(char));
	}
	hipMemcpy(terrain_player_sight_device, terrain_player_sight_temp, player_sight_size * sizeof(char*), hipMemcpyHostToDevice);
	for (int i = 0; i < player_sight_size; i++) {
		hipMemcpy(terrain_player_sight_temp[i], terrain_player_sight_host[i], player_sight_size * sizeof(char), hipMemcpyHostToDevice);
	}

	
	//Terrain move & Player Sight Update===================================================
	TI player_location = {0, 0};		//이거 나중에 중심 기준으로 바꿔야함
	int wind_angle = 270;		//각도
	int wind_speed = 50;		//최대 풍속 50
	for (int i = 0; i < 10; i++) {
		clock_t t_1 = clock();

		//Terrain Move
		wind_decide(wind_speed, wind_angle);

		TF wind_direction = { cos(wind_angle * PI / 180), sin(wind_angle * PI / 180) };
		if (abs(wind_direction.x) < FLT_EPSILON) {
			wind_direction.x = 0;
		}
		if (abs(wind_direction.y) < FLT_EPSILON) {
			wind_direction.y = 0;
		}
		
		move_terrain(hill_location_host, num_of_hills, wind_direction, wind_speed);
		if (num_of_hills < origin_num_of_hills) {
			make_new_hills(hill_location_host, num_of_hills, origin_num_of_hills, wind_direction, wind_speed);
		}
		
		hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device

		//Player Sight Update
		//player_location.x += 20;
		//player_location.y += 20;
		//thread must be 1024 for efficiency
		player_terrain_update_cuda <<<player_sight_size, player_sight_size >>> (terrain_player_sight_device, hill_location_device, num_of_hills, player_location, wind_direction , wind_speed);
		for (int i = 0; i < player_sight_size; i++) {
			hipMemcpy(terrain_player_sight_host[i], terrain_player_sight_temp[i], player_sight_size * sizeof(char), hipMemcpyDeviceToHost);
		}
		clock_t t_2 = clock();
		cout << "Player Sight Update Time : " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " Seconds" << endl;
		//show_array(terrain_player_sight_host, player_sight_size);
		cout << "==============================" << endl;
	}
	
	
	
	//File Save===================================================
	/*ofstream out{"terrain.txt"};

	for (int i = 0; i < one_side_number; i++) {
		for (int j = 0; j < one_side_number; j++) {
			out << terrain_array_host[i][j] << " ";
		}
		out << endl;
	}
	cout << "File Write Complete" << endl;*/
	

	//Free Memory===================================================
	delete[] hill_location_host;
	hipFree(hill_location_device);
	for (int i = 0; i < player_sight_size; i++) {
		delete[] terrain_player_sight_host[i];
		hipFree(terrain_player_sight_temp[i]);
	}
	delete[] terrain_player_sight_host;
	hipFree(terrain_player_sight_device);
	hipFree(terrain_player_sight_temp);
}
