#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <vector>
#include <random>
#include <windows.h>
#include <time.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include ""
using namespace std;

const unsigned int one_side_number = 1000;	//39936
const int player_sight_size = 50;	//1024 넘으면 안됨

const int max_height = 8;
const int base_floor = 1;

typedef struct two_int {
	int x;
	int y;
} TI;

typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>hills_location(0, one_side_number);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 10);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(5, max_height);

void get_device_info()
{
	hipDeviceProp_t  prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
		printf("\n");

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}

void show_array(char** terrain_array_host, int size)
{
	for (int x = 0; x < size; x++) {
		for (int y = 0; y < size; y++) {
			printf("%d ", terrain_array_host[x][y]);
		}
		printf("\n");
	}
}

__global__
void make_hills_cuda(char** terrain_array_device, HI* hill_location_device)
{
	//int id = threadIdx.x + blockIdx.x * blockDim.x;
	int my_hill = threadIdx.x;
	int my_y = blockIdx.x;

	int hill_location_x = hill_location_device[my_hill].x;
	int hill_location_y = hill_location_device[my_hill].y;
	int radius = hill_location_device[my_hill].radius;
	int height = hill_location_device[my_hill].height;
	int distance{};

	//printf("threadId.x: %d, blockIdx.x: %d, blockDim.x: %d, id: %d\n", threadIdx.x, blockIdx.x, blockDim.x, id);
	//printf("Hill %d : (%d, %d) / %d / %d\n", my_hill, hill_location_x, hill_location_y, radius, height);

	for (int x = hill_location_x - radius; x <= hill_location_x + radius; ++x) {
		distance = sqrt(((pow(x - hill_location_x, 2)) + (pow(my_y - hill_location_y, 2))));
		if (distance <= radius) {
			terrain_array_device[x][my_y] += (height - 1) * (radius - distance) / radius + 1;
			if (terrain_array_device[x][my_y] > max_height) {
				terrain_array_device[x][my_y] = max_height;
			}
			else if (terrain_array_device[x][my_y] < 0) {
				terrain_array_device[x][my_y] = 0;
			}
		}
	}
}

int make_hill_location(HI* hill_location_host)
{
	int num_of_hills = number_of_hills_uid(dre);

	cout << "expected num of hills: " << num_of_hills << endl;

	for (int i = 0; i < num_of_hills; i++) {
		hill_location_host[i].x = hills_location(dre);
		hill_location_host[i].y = hills_location(dre);
		hill_location_host[i].radius = hill_size_uid(dre);
		hill_location_host[i].height = height_uid(dre);
	}
	sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
	for (int a = 0; a < num_of_hills; a++) {
		for (int b = 0; b < num_of_hills; b++) {
			if (a != b) {
				if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
					if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						for (int i = b; i < num_of_hills; i++) {
							hill_location_host[i] = hill_location_host[i + 1];
						}
						b -= 1;
						num_of_hills -= 1;
					}
				}
			}
		}
	}
	for (int i = 0; i < num_of_hills; i++) {
		if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
	}
	cout << "real num of hills: " << num_of_hills << endl;
	return num_of_hills;
}

__global__
void player_terrain_update_cuda(char** terrain_array_device, char** terrain_player_sight_device, TI player_location, int player_sight_size)
{
	//int id = threadIdx.x + blockIdx.x * blockDim.x;
	int x = threadIdx.x;
	int y = blockIdx.x;
	//printf("threadId.x: %d, blockIdx.x: %d, blockDim.x: %d\n", threadIdx.x, blockIdx.x, blockDim.x);

	if (x + player_location.x > 0 && x + player_location.x < one_side_number && y + player_location.y > 0 && y + player_location.y < one_side_number) {
		terrain_player_sight_device[x][y] = terrain_array_device[x + player_location.x][y + player_location.y];
	}
	else {
		terrain_player_sight_device[x][y] = 0;
	}
}

int main()
{
	//ToDo
	//마을 위치 랜덤 생성, 마을 위치는 높이 0인데 주변 언덕이 점차적인 높이로함, 언덕 움직이기
	// 언덕을 움직이는데 효울적인 방법: 플레이어가 보는 시야만 업데이트를 하게끔 해야함 
	// hill_location 업데이트는 항상 하고, cuda로 플레이어가 보는 시점만 terrain_array를 업데이트 해야함
	// 플레이어가 보는 시점을 1000*1000이라 가정하고, 그 부분의 terrain_array를 업데이트 해야함
	// 플레이어 수에 맞게 플레이어 시점 2차원 배열 terrain_array_for_player를 동적할당하여 생성해야함
	// 
	//
	
	//get_device_info();

	//Make Random Hills Information===================================================
	clock_t random_hills_info_start_time = clock();
	HI* hill_location_host = new HI[10000];
	HI* hill_location_device;
	hipMalloc((void**)&hill_location_device, 10000 * sizeof(HI));
	int num_of_hills = make_hill_location(hill_location_host);
	hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device
	printf("Random Hill Info Complete\n");
	
		
	//Terrain Memory Assignement===================================================
	clock_t memory_assign_start_time = clock();
	char** terrain_array_host = new char* [one_side_number];	// 2D array for host
	for (int i = 0; i < one_side_number; i++) {
		terrain_array_host[i] = new char[one_side_number];
	}
	for (int i = 0; i < one_side_number; i++) {
		for (int j = 0; j < one_side_number; j++) {
			terrain_array_host[i][j] = 1;
		}
	}
	char** terrain_array_device;					// 2D array for device
	char* terrain_array_temp[one_side_number];		// 1D array temp
	hipMalloc((void**)&terrain_array_device, one_side_number * sizeof(char*));
	for (int i = 0; i < one_side_number; i++) {
		hipMalloc((void**)&terrain_array_temp[i], one_side_number * sizeof(char));
	}
	hipMemcpy(terrain_array_device, terrain_array_temp, one_side_number* sizeof(char*), hipMemcpyHostToDevice);
	for (int i = 0; i < one_side_number; i++) {
		hipMemcpy(terrain_array_temp[i], terrain_array_host[i], one_side_number * sizeof(char), hipMemcpyHostToDevice);
	}
	
	
	//Make Hills===================================================
	clock_t terrain_generate_start_time = clock();
	make_hills_cuda << <one_side_number, num_of_hills >> > (terrain_array_device, hill_location_device);
	for (int i = 0; i < one_side_number; i++) {	
		hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
	}
	printf("Terrain Generation Complete\n");
	clock_t  terrain_generate_end_time = clock();

	//show_array(terrain_array_host, one_side_number);
	cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
	cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
	cout << "Random Hills Info Time : " << (double)(memory_assign_start_time - random_hills_info_start_time) / CLOCKS_PER_SEC << " sec" << endl;
	cout << "Memory Assign Time : " << (double)(terrain_generate_start_time - memory_assign_start_time) / CLOCKS_PER_SEC << " sec" << endl;
	cout << "Terrain Generate Time : " << (double)(terrain_generate_end_time - terrain_generate_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
	cout << "Total Time : " << (double)(terrain_generate_end_time - random_hills_info_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;

	
	//Terrain Memory Assignment For Player's Sight===================================================
	char** terrain_player_sight_host = new char* [player_sight_size];	// 2D array for host
	for (int i = 0; i < player_sight_size; i++) {
		terrain_player_sight_host[i] = new char[player_sight_size];
	}
	for (int i = 0; i < player_sight_size; i++) {
		for (int j = 0; j < player_sight_size; j++) {
			terrain_player_sight_host[i][j] = 0;
		}
	}
	char** terrain_player_sight_device;					// 2D array for device
	char* terrain_player_sight_temp[player_sight_size];		// 1D array temp
	hipMalloc((void**)&terrain_player_sight_device, player_sight_size * sizeof(char*));
	for (int i = 0; i < player_sight_size; i++) {
		hipMalloc((void**)&terrain_player_sight_temp[i], player_sight_size * sizeof(char));
	}
	hipMemcpy(terrain_player_sight_device, terrain_player_sight_temp, player_sight_size * sizeof(char*), hipMemcpyHostToDevice);
	for (int i = 0; i < player_sight_size; i++) {
		hipMemcpy(terrain_player_sight_temp[i], terrain_player_sight_host[i], player_sight_size * sizeof(char), hipMemcpyHostToDevice);
	}

	
	//Player Sight Update===================================================
	TI player_location = { 1,500 };

	for (int i = 0; i < 10; i++) {
		clock_t player_sight_update_start_time = clock();
		player_location.x += 50;
		player_location.y += 50;
		//thread 1024 넘으면 문제 안생기나?
		player_terrain_update_cuda << <player_sight_size, player_sight_size >> > (terrain_array_device, terrain_player_sight_device, player_location, player_sight_size);
		for (int i = 0; i < player_sight_size; i++) {
			hipMemcpy(terrain_player_sight_host[i], terrain_player_sight_temp[i], player_sight_size * sizeof(char), hipMemcpyDeviceToHost);
		}
		clock_t player_sight_update_end_time = clock();
		cout << "Player Sight Update Time : " << (double)(player_sight_update_end_time - player_sight_update_start_time) / CLOCKS_PER_SEC << " Seconds" << endl;
		show_array(terrain_player_sight_host, player_sight_size);
		cout << "==============================" << endl;
	}
	
	
	//File Save===================================================
	/*ofstream out{"terrain.txt"};

	for (int i = 0; i < one_side_number; i++) {
		for (int j = 0; j < one_side_number; j++) {
			out << terrain_array_host[i][j] << " ";
		}
		out << endl;
	}
	cout << "File Write Complete" << endl;*/
	

	//Free Memory===================================================
	delete hill_location_host;
	hipFree(hill_location_device);
	
	for (int i = 0; i < one_side_number; i++) {
		delete[] terrain_array_host[i];
	}
	delete[] terrain_array_host;
	hipFree(terrain_array_device);
	for (int i = 0; i < one_side_number; i++) {
		hipFree(terrain_array_temp[i]);
	}
}
