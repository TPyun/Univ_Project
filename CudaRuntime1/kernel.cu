#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <random>
#include <windows.h>
#include <time.h>
#include <math.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include ""
#define PI 3.1415926
using namespace std;

const unsigned int one_side_number = 500;	//39936
const int player_sight_size = 500;	//1024 넘으면 안됨

const int max_height = 8;
const int base_floor = 1;

typedef struct two_int {
	int x;
	int y;
} TI;

typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>hills_location(0, one_side_number);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 10);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(5, max_height);

void get_device_info()
{
	hipDeviceProp_t  prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
		printf("\n");

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}

void show_array(char** terrain_array_host, int size)
{
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			printf("%d ", terrain_array_host[x][y]);
		}
		printf("\n");
	}
}

int make_hill_location(HI* hill_location_host)
{
	int num_of_hills = number_of_hills_uid(dre);

	cout << "expected num of hills: " << num_of_hills << endl;

	for (int i = 0; i < num_of_hills; i++) {
		hill_location_host[i].x = hills_location(dre);
		hill_location_host[i].y = hills_location(dre);
		hill_location_host[i].radius = hill_size_uid(dre);
		hill_location_host[i].height = height_uid(dre);
	}
	sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
	for (int a = 0; a < num_of_hills; a++) {
		for (int b = 0; b < num_of_hills; b++) {
			if (a != b) {
				if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
					if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						for (int i = b; i < num_of_hills; i++) {
							hill_location_host[i] = hill_location_host[i + 1];
						}
						b -= 1;
						num_of_hills -= 1;
					}
				}
			}
		}
	}
	for (int i = 0; i < num_of_hills; i++) {
		if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
		else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
			for (int j = i; j < num_of_hills; j++) {
				hill_location_host[j] = hill_location_host[j + 1];
			}
			i -= 1;
			num_of_hills -= 1;
		}
	}
	cout << "real num of hills: " << num_of_hills << endl;
	return num_of_hills;
}

void move_terrain(HI* hill_location_host, int num_of_hills, int wind_direction, int wind_speed)
{
	if(wind_direction)
	for (int i = 0; i < num_of_hills; i++) {
		hill_location_host[i].x += wind_speed * cos(wind_direction * PI / 180);
		hill_location_host[i].y += wind_speed * sin(wind_direction * PI / 180);
	}
}

__global__
void player_terrain_update_cuda(char** terrain_player_sight_device, HI* hill_location_device, int num_of_hills, TI player_location, int wind_direction)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int terrain_x = player_location.x + x;
	int terrain_y = player_location.y + y;
	
	if (terrain_x >= 0 && terrain_x <= one_side_number && terrain_y >= 0 && terrain_y <= one_side_number) {
		terrain_player_sight_device[x][y] = base_floor;
		for (int i = 0; i < num_of_hills; i++) {
			
			//원래 언덕 채우기
			int hill_location_x = hill_location_device[i].x;
			int hill_location_y = hill_location_device[i].y;
			int radius = hill_location_device[i].radius;
			int height = hill_location_device[i].height;
			int distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

			if (distance <= radius) {
				terrain_player_sight_device[x][y] += (height) * (radius - distance) / radius;

				//언덕 깎기
				hill_location_x = hill_location_device[i].x - distance * cosf(wind_direction * PI / 180);
				hill_location_y = hill_location_device[i].y - distance * sinf(wind_direction * PI / 180);
				distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

				if (distance <= radius) {
					terrain_player_sight_device[x][y] -= height * (radius - distance) / radius;
					if (terrain_player_sight_device[x][y] > max_height) {
						terrain_player_sight_device[x][y] = max_height;
					}
					else if (terrain_player_sight_device[x][y] < base_floor) {
						terrain_player_sight_device[x][y] = base_floor;
					}
				}
			}
		}
	}
	else {
		//맵 밖의 경우 0
		terrain_player_sight_device[x][y] = 0;
	}
}

int main()
{
	//get_device_info();

	
	//ToDo
	//마을 위치 랜덤 생성, 마을 위치는 높이 0인데 주변 언덕이 점차적인 높이로함, 언덕 움직이기
	// 언덕을 움직이는데 효울적인 방법: 플레이어가 보는 시야만 업데이트를 하게끔 해야함 
	// hill_location 업데이트는 항상 하고, cuda로 플레이어가 보는 시점만 terrain_array를 업데이트 해야함
	// 플레이어가 보는 시점을 1000*1000이라 가정하고, 그 부분의 terrain_array를 업데이트 해야함
	// 플레이어 수에 맞게 플레이어 시점 2차원 배열 terrain_array_for_player를 동적할당하여 생성해야함
	// 
	// 플레이어에게 맵 정보를 보내줘야 할 시기: 게임 시작, 지형이 변경될 때, 카메라가 움직일 때
	// 문제: 지형이 움직일때 terrain_array전체를 수정하고, 플레이어에게 terrain_array에 복사만 해서 일부분을 보여줄지,
	// hill_location을 수정하고, 플레이어에게 보여줄때, terrain_array를 아예 그릴지. ====이게 나을듯...40000*40000 만드는건 뻘짓이였다..
	//
	// 지형이 움직이면, 움직이는 방향의 반대 부분은 비게 된다. 언덕이 없다던가 등등...
	// 지형이 한 방향으로 움직이면 생성해줘야함
	// 그리고 40000 * 40000 밖으로 나간 언덕은  hill_location 배열에서 지워줘야함.
	// 
	// 언덕이 생성될때 언덕의 가파른 정도를 생각해보는 것도 좋을듯 하다
	// 
	// 사막의 모래언덕 모양을 위에서 보게 된다면 상현달~초승달의 모양을 띄게 되는것같다
	// 바람의 세기와 방향에 따라서 달 모양이 향하는 방향을 바꿔줘야한다.
	// 바람이 한번도 불지 않았다면 원 모양을 띈다는 전제하에 바람이 불면, 바람이 온 방향쪽을 깎아주는 모양을 만들자
	// 바람이 서쪽에서 분다면 언덕의 왼쪽 부분을 깎고 언덕이 오른쪽으로 이동하게끔 한다,
	// 그 후 바람이 동쪽에서 분다면 한칸씩 왼쪽은 차게되고, 오른쪽은 깎이는 형식으로 만들자.
	// 바람의 방향이 끼치는 영향: 언덕이 깎였던 부분이 채워지는 방향, 언덕이 깎이는 방향
	// 바람의 세기가 끼치는 영향: 지형이 이동하는 속도, 언덕이 채워지는 속도, 언덕이 깎이는 속도
	// 
	// 블럭의 가로 세로 높이 비율을 1:1:0.2로 하면 어떨까 생각이 듦.
	// 만약 1:1:1이라면 붙어있는 블럭의 경사는 0, 45도 밖에 없음
	// 
	//
	

	//Make Random Hills Information===================================================
	HI* hill_location_host = new HI[4000];
	HI* hill_location_device;
	hipMalloc((void**)&hill_location_device, 4000 * sizeof(HI));
	int num_of_hills = make_hill_location(hill_location_host);
	hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device
	printf("Random Hill Info Complete\n");

	
	//Terrain Memory Assignment For Player's Sight===================================================
	char** terrain_player_sight_host = new char* [player_sight_size];	// 2D array for host
	for (int i = 0; i < player_sight_size; i++) {
		terrain_player_sight_host[i] = new char[player_sight_size];
	}
	for (int i = 0; i < player_sight_size; i++) {
		for (int j = 0; j < player_sight_size; j++) {
			terrain_player_sight_host[i][j] = 0;
		}
	}
	char** terrain_player_sight_device;					// 2D array for device
	char* terrain_player_sight_temp[player_sight_size];		// 1D array temp
	hipMalloc((void**)&terrain_player_sight_device, player_sight_size * sizeof(char*));
	for (int i = 0; i < player_sight_size; i++) {
		hipMalloc((void**)&terrain_player_sight_temp[i], player_sight_size * sizeof(char));
	}
	hipMemcpy(terrain_player_sight_device, terrain_player_sight_temp, player_sight_size * sizeof(char*), hipMemcpyHostToDevice);
	for (int i = 0; i < player_sight_size; i++) {
		hipMemcpy(terrain_player_sight_temp[i], terrain_player_sight_host[i], player_sight_size * sizeof(char), hipMemcpyHostToDevice);
	}

	
	//Terrain move & Player Sight Update===================================================
	TI player_location = {0, 0};
	for (int i = 0; i < 10; i++) {
		clock_t t_1 = clock();
		//Terrain Move
		int wind_direction = 70;	//각도
		int wind_speed = 100;
		move_terrain(hill_location_host, num_of_hills, wind_direction, wind_speed);
		hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device

		//Player Sight Update
		//player_location.x += 20;
		//player_location.y += 20;
		//thread must be 1024 for efficiency
		player_terrain_update_cuda <<<player_sight_size, player_sight_size >>> (terrain_player_sight_device, hill_location_device, num_of_hills, player_location, wind_direction);
		for (int i = 0; i < player_sight_size; i++) {
			hipMemcpy(terrain_player_sight_host[i], terrain_player_sight_temp[i], player_sight_size * sizeof(char), hipMemcpyDeviceToHost);
		}
		clock_t t_2 = clock();
		cout << "Player Sight Update Time : " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " Seconds" << endl;
		show_array(terrain_player_sight_host, player_sight_size);
		cout << "==============================" << endl;
	}
	
	
	
	//File Save===================================================
	/*ofstream out{"terrain.txt"};

	for (int i = 0; i < one_side_number; i++) {
		for (int j = 0; j < one_side_number; j++) {
			out << terrain_array_host[i][j] << " ";
		}
		out << endl;
	}
	cout << "File Write Complete" << endl;*/
	

	//Free Memory===================================================
	delete[] hill_location_host;
	hipFree(hill_location_device);
	for (int i = 0; i < player_sight_size; i++) {
		delete[] terrain_player_sight_host[i];
		hipFree(terrain_player_sight_temp[i]);
	}
	delete[] terrain_player_sight_host;
	hipFree(terrain_player_sight_device);
	hipFree(terrain_player_sight_temp);
}
