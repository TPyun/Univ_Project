#include "hip/hip_runtime.h"
#include <stdio.h>
#include "\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.0\include\hip/hip_runtime.h"

void helloCPU()
{
    printf("Hello from the CPU.\n");
}

__global__ void helloGPU()
{
    for (int i = 0; i < 40000; ++i) {
        for (int j = 0; j < 40000; ++j) {
            printf("%d ", j);
        }
        printf("\n");
    }
}
