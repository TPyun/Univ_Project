#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <random>
#include <windows.h>
#include <time.h>
#include <math.h>
#include <fstream>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include ""
#include "global.h"
#define PI 3.1415926
using namespace std;

//const int one_side_number = 640;	//32000
//const int player_sight_size = 50;	//1024 ������ �ȵ�
//const int random_array_size = 500000;// 90000000;
const int one_side_number = 16000;	//32000
const int player_sight_size = 70;	//1024 ������ �ȵ�
const int random_array_size = 50000000;// 90000000;

const int max_height = 8;
const int base_floor = 1;

typedef struct two_int {
	int x;
	int y;
} II;
typedef struct two_char {
	int x;
	int y;
} CC;
typedef struct two_float {
	float x;
	float y;
} FF;
typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;
typedef struct compare_block {
	II middle;
	II up;
	II down;
	II left;
	II right;
}CB;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>terrain_distance(0, one_side_number - 1);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 10);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(4, max_height);

uniform_int_distribution <int>wind_speed_uid(0, 50);
uniform_int_distribution <int>wind_angle_uid(0, 360);

void make_random_array(II* random_array, bool& random_array_used)
{
	while (1) {
		if (random_array_used) {
			clock_t t_0 = clock();
			for (int i = 0; i < random_array_size; i++) {
				random_array[i].x = terrain_distance(dre);
				random_array[i].y = terrain_distance(dre);
			}
			random_array_used = false;
			clock_t t_1 = clock();
			//cout << "_Thread_ Random for Scarce: " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec\n";
		}
		else {
			Sleep(10);
		}
	}
}

__global__
void make_hills_cuda(char** terrain_array_device, HI* hill_location_device, int num_of_hills)
{
	II terrain;
	terrain.x = blockIdx.x * blockDim.x + threadIdx.x;
	terrain.y = blockIdx.y * blockDim.y + threadIdx.y;
	HI hill;
	int distance;

	if (terrain.x % 5 == 0 && terrain.y % 5 == 0) {
		terrain_array_device[terrain.x][terrain.y] += 4;
	}
}

__global__
void terrain_corrosion_cuda(char** terrain_array_device, int height)
{
	//�ٶ��� �Ҹ� �������� �̵��ϰԲ� �����ؾ� ��
	//�ƴϸ� �ٶ����� ���� �Լ� ���� ���� �ߺ����� ���� �� ��� Ȯ��
	const int block_num = 9;
	II terrain[block_num];
	terrain[0].x = blockIdx.x * blockDim.x + threadIdx.x;	//middle
	terrain[0].y = blockIdx.y * blockDim.y + threadIdx.y;

	if (terrain_array_device[terrain[0].x][terrain[0].y] != height) {	//���ڷ� ���� ���̰� �ƴϸ� ����
		return;
	}
	if (terrain_array_device[terrain[0].x][terrain[0].y] <= base_floor) {	//base floor ���� ������ ���̻� ���߸� �ȵ�
		return;
	}
	
	terrain[1] = { terrain[0].x, terrain[0].y - 1 };		//up
	terrain[2] = { terrain[0].x, terrain[0].y + 1 };		//down
	terrain[3] = { terrain[0].x - 1, terrain[0].y };		//left
	terrain[4] = { terrain[0].x + 1, terrain[0].y };		//right
	terrain[5] = { terrain[0].x - 1, terrain[0].y - 1 };	//up left
	terrain[6] = { terrain[0].x - 1, terrain[0].y + 1 };	//down left
	terrain[7] = { terrain[0].x + 1, terrain[0].y - 1 };	//up right
	terrain[8] = { terrain[0].x + 1, terrain[0].y + 1 };	//down right

	//�����¿� ����
	for (int i = 0; i < block_num; i++) {
		if (terrain[i].x < 0) {
			terrain[i].x += one_side_number;
		}
		if (terrain[i].x >= one_side_number) {
			terrain[i].x -= one_side_number;
		}
		if (terrain[i].y < 0) {
			terrain[i].y += one_side_number;
		}
		if (terrain[i].y >= one_side_number) {
			terrain[i].y -= one_side_number;
		}
	}

	int height_difference = 0;
	for (int i = 1; i < block_num; i++) {	//�ֺ� ������ ���� ���̸� ����
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] > height_difference) {
			height_difference = terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y];
		}
	}

	if (height_difference <= 0) {	//�ֺ��� �� ���� ���� ������ ����
		return;
	}

	int num_of_lowest{};
	for (int i = 1; i < block_num; i++) {
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
			num_of_lowest++;
		}
	}

	if (num_of_lowest == 1) {	//���� ���� ���̰� �ϳ��� �ű�� �̵�
		for (int i = 1; i < block_num; i++) {
			if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
				terrain_array_device[terrain[i].x][terrain[i].y]++;
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
		}
	}
	if (num_of_lowest == 8 && height_difference == 1) {	//�ֺ��� ��� ���̰� ���� ���� ���̰� 1�̸� ����
		return;
	}
	
	char thread_seed = (terrain[0].x + terrain[0].y + abs(terrain[0].x + terrain[0].y * height)) % num_of_lowest;	//random seed made by myself
	int iter = 0;
	for (int i = 0; i < block_num; i++) {
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {	//���� ���� ���̰� �������� �������� �ϳ� ����
			if (thread_seed == iter) {	//�������� ���õ� ������ �̵�
				terrain_array_device[terrain[i].x][terrain[i].y]++;
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
			iter++;
		}
	}
}

__global__
void wind_blow_cuda(char** terrain_array_device, II wind_direction)
{
	//wind_direction�� x, y�� �ϳ��� ������ 0�̿��� ��
	II terrain[5];
	terrain[0].x = blockIdx.y * blockDim.y + threadIdx.y;	//����ϰԵ� x,y�°� �ϸ� ���� ���� ��ȭ�ҽ� Ư�� ���̰� ����. �̷��� �ϸ� �Ȼ���. �̷��� �ٲ㵵 �������� ���� ��
	terrain[0].y = blockIdx.x * blockDim.x + threadIdx.x;	//middle

	terrain[1].x = terrain[0].x + wind_direction.x;			//forward
	terrain[1].y = terrain[0].y + wind_direction.y;

	terrain[2].x = terrain[1].x + wind_direction.y;			//forward left
	terrain[2].y = terrain[1].y - wind_direction.x;
	
	terrain[3].x = terrain[1].x - wind_direction.y;			//forward right
	terrain[3].y = terrain[1].y + wind_direction.x;

	terrain[4].x = terrain[0].x - wind_direction.x;			//back
	terrain[4].y = terrain[0].y - wind_direction.y;
	
	//���� �����¿� ����
	for (int i = 0; i < 5; i++) 
	{
		if (terrain[i].x < 0 ) {
			terrain[i].x += one_side_number;
		}
		if (terrain[i].x >= one_side_number) {
			terrain[i].x -= one_side_number;
		}
		if (terrain[i].y < 0) {
			terrain[i].y += one_side_number;
		}
		if (terrain[i].y >= one_side_number) {
			terrain[i].y -= one_side_number;
		}
	}

	if (terrain_array_device[terrain[0].x][terrain[0].y] <= base_floor) 
	{	//base floor ���� ������ ���̻� ���߸� �ȵ�
		return;
	}
	//�̰Ŷ����� ������ ������
	if (terrain_array_device[terrain[0].x][terrain[0].y] > base_floor && terrain_array_device[terrain[0].x][terrain[0].y] == terrain_array_device[terrain[4].x][terrain[4].y] && terrain_array_device[terrain[0].x][terrain[0].y] == terrain_array_device[terrain[1].x][terrain[1].y]) {	//��� �þ�� �������� ����
		
	}
	else if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[4].x][terrain[4].y] <= 0) 
	{	//�Ĺ� ���� �� ������ ���ų� ���� ��� �ٶ��� ������ ���� ����
		return;
	}

	int height_difference = -1;
	for (int i = 1; i < 4; i++) 
	{
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y]  > height_difference) 
		{
			height_difference = terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y];	//i���� 0���� ���� ����
		}
	}
	if (height_difference < 0) {	//���� 3�� ������ ���̰� ��� �� ������ ���� ��� ���� �ȹ���
		return;
	}

	int num_of_lowest{};
	for (int i = 1; i < 4; i++) 
	{
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) 
		{	//���� ���� ���� �����
			num_of_lowest++;
		}
	}

	if (num_of_lowest == 0)
		return; 
	
	//�������̰� �ϳ��� ��� �׸��� ��
	if (num_of_lowest == 1) 
	{	
		for (int i = 1; i < 4; i++) 
		{
			if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) 
			{
				/*atomic_fetch_add(&terrain_array_device[terrain[i].x][terrain[i].y], -1);
				atomic_fetch_add(&terrain_array_device[terrain[0].x][terrain[0].y], 1);*/
				--terrain_array_device[terrain[0].x][terrain[0].y];
				++terrain_array_device[terrain[i].x][terrain[i].y];
				return;
			}
		}
	}
	
	//�������̰� ������ �� ��, ���� ���� ���� �����̸� �������� ����
	if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[1].x][terrain[1].y] == height_difference) 
	{		
		/*atomicAdd(&terrain_array_device[terrain[1].x][terrain[1].y], -1);
		atomicAdd(&terrain_array_device[terrain[0].x][terrain[0].y], 1);*/
		--terrain_array_device[terrain[0].x][terrain[0].y];
		++terrain_array_device[terrain[1].x][terrain[1].y];
		return;
	}

	int radom_seed = (terrain[0].x + terrain[0].y) % num_of_lowest + 2;
	//�� �� ���� �ϳ��� ���� �̵�
	/*atomicAdd(&terrain_array_device[terrain[radom_seed].x][terrain[radom_seed].y], -1);
	atomicAdd(&terrain_array_device[terrain[0].x][terrain[0].y], 1);*/
	--terrain_array_device[terrain[0].x][terrain[0].y];
	++terrain_array_device[terrain[radom_seed].x][terrain[radom_seed].y];
	return;
}

__global__
void add_scarce_cuda(char** terrain_array_device, II* random_seed_device, int size) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	II my_seed;
	my_seed.x = random_seed_device[id].x;
	my_seed.y = random_seed_device[id].y;

	terrain_array_device[my_seed.x][my_seed.y]++;
}

__global__
void player_terrain_update_cuda(char** terrain_player_sight_device, HI* hill_location_device, int num_of_hills, II player_location, FF wind_direction, int wind_speed)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int terrain_x = player_location.x + x;
	int terrain_y = player_location.y + y;

	if (terrain_x >= 0 && terrain_x <= one_side_number && terrain_y >= 0 && terrain_y <= one_side_number) {
		terrain_player_sight_device[x][y] = base_floor;
		for (int i = 0; i < num_of_hills; i++) {
			
			//���� ��� ä���
			int hill_location_x = hill_location_device[i].x;
			int hill_location_y = hill_location_device[i].y;
			int radius = hill_location_device[i].radius;
			int height = hill_location_device[i].height;
			int distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

			if (distance <= radius) {
				terrain_player_sight_device[x][y] += (height) * (radius - distance) / radius;

				//��� ���
				hill_location_x = hill_location_device[i].x - radius * wind_direction.x * (100 - wind_speed) / 50;
				hill_location_y = hill_location_device[i].y - radius * wind_direction.y * (100 - wind_speed) / 50;
				distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

				if (distance <= radius) {
					terrain_player_sight_device[x][y] -= height * (radius - distance) / radius + base_floor;
					if (terrain_player_sight_device[x][y] > max_height) {
						terrain_player_sight_device[x][y] = max_height;
					}
					else if (terrain_player_sight_device[x][y] < base_floor) {
						terrain_player_sight_device[x][y] = base_floor;
					}
				}
			}
		}
	}
	else {
		//�� ���� ��� 0
		terrain_player_sight_device[x][y] = 0;
	}
}

__global__
void except_city_terrain_cuda(char** terrain_array_device, II* city_location_device, int num_of_city)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int inner_radius = 20;
	int outer_radius = inner_radius + 20;
	II distance;

	for (int i = 0; i < num_of_city; i++) {
		if (city_location_device[i].x == 0 && city_location_device[i].y == 0) 
			continue;
		if (abs(x - city_location_device[i].x) <= inner_radius && abs(y - city_location_device[i].y) <= inner_radius) {
			terrain_array_device[x][y] = 1;
			return;
		}
		
		distance.x = abs(x - city_location_device[i].x);
		distance.y = abs(y - city_location_device[i].y);
		//printf("%d, %d\n", distance.x, distance.y);
		if (distance.x > outer_radius || distance.y > outer_radius)
			continue;
		
		if (distance.x == distance.y && distance.x == 0) 
			terrain_array_device[x][y] -= outer_radius;
		else if (distance.x >= distance.y) 
			terrain_array_device[x][y] += distance.x - outer_radius;
		else if (distance.x < distance.y) 
			terrain_array_device[x][y] += distance.y - outer_radius;
		
		if (terrain_array_device[x][y] < base_floor)
			terrain_array_device[x][y] = base_floor;
	}
}

__global__
void make_shadow_map_cuda(char** terrain_array_device, char** shadow_map_device, int sun_angle)
{
	II coo;
	coo.y = blockIdx.y * blockDim.y + threadIdx.y;
	coo.x = blockIdx.x * blockDim.x + threadIdx.x;
	
	int height = terrain_array_device[coo.x][coo.y];
	float ratio = tanf(sun_angle * PI / 180);

	if (abs(ratio) < FLT_EPSILON) {
		ratio = 0;
	}
	int distance = height / ratio * -1;

	if (distance < 0) {
		for (int i = -1; i >= distance; i--) {
			//printf("%d, %d\n", i, distance);
			if (coo.x + i < 0) {
				break;
			}
			if (terrain_array_device[coo.x + i][coo.y] >= height) {
				break;
			}
			shadow_map_device[coo.x + i][coo.y] = 0;
		}
	}
	else if (distance > 0) {
		for (int i = 1; i <= distance; i++) {
			if (coo.x + i >= one_side_number) {
				break;
			}
			if (terrain_array_device[coo.x + i][coo.y] >= height) {
				break;
			}
			shadow_map_device[coo.x + i][coo.y] = 0;
		}
	}
}

__global__
void make_temperature_map_cuda(char** terrain_array_device, char** shadow_map_device, char** temperature_map_device, int sun_angle)
{
	II coo;
	coo.x = blockIdx.y * blockDim.y + threadIdx.y;
	coo.y = blockIdx.x * blockDim.x + threadIdx.x;

	char height[3];
	height[1] = terrain_array_device[coo.x][coo.y];	//�߽�

	if (coo.x - 1 < 0) {
		height[0] = height[1];
	}
	else {
		height[0] = terrain_array_device[coo.x - 1][coo.y];
	}
	if (coo.x + 1 >= one_side_number)
	{
		height[2] = height[1];
	}
	else {
		height[2] = terrain_array_device[coo.x + 1][coo.y];
	}

	if (shadow_map_device[coo.x][coo.y] == 1) {
		int ground_angle{};
		if (height[0] > height[2]) {
			ground_angle = (atanf(abs(height[0] - height[2])) * 180 / PI);
		}
		else if (height[0] < height[2]) {
			ground_angle = (atanf(abs(height[0] - height[2])) * 180 / PI) + 90;
		}
		else {
			ground_angle = 90;
		}
		// �µ� ��ȯ�� ����� ������ ��.
		// sun angle ������Ʈ�Ҷ����� �µ� ������Ʈ ���� �µ� ������Ʈ �ð��� �����ɸ�
		// �Ϸ� 3�п� �°� sunangle ������Ʈ �ؾ���
		int angle_difference = (180 - abs(ground_angle - sun_angle));
		int temperature = angle_difference / 10;
		temperature_map_device[coo.x][coo.y] += angle_difference;
		if (angle_difference > 0) {
			//printf("%d %d\n", angle_difference, temperature);
		}
	}
	else if (shadow_map_device[coo.x][coo.y] == 0) {
		//temperature_map_device[coo.x][coo.y] -= 1;
	}
}

__global__
void heat_conduction_cuda(char** temperature_map_device)
{
	II coo;
	coo.x = blockIdx.y * blockDim.y + threadIdx.y;
	coo.y = blockIdx.x * blockDim.x + threadIdx.x;

	char temperature[9];
	temperature[4] = temperature_map_device[coo.x][coo.y];	//�߽�
	if (coo.x - 1 >= 0) {
		temperature[0] = temperature_map_device[coo.x - 1][coo.y - 1];
		temperature[3] = temperature_map_device[coo.x - 1][coo.y];
		temperature[6] = temperature_map_device[coo.x - 1][coo.y + 1];
	}
	else {
		temperature[0] = temperature[4];
		temperature[3] = temperature[4];
		temperature[6] = temperature[4];
	}
	
	if (coo.x + 1 < one_side_number){
		temperature[2] = temperature_map_device[coo.x + 1][coo.y - 1];
		temperature[5] = temperature_map_device[coo.x + 1][coo.y];
		temperature[8] = temperature_map_device[coo.x + 1][coo.y + 1];
	}
	else {
		temperature[2] = temperature[4];
		temperature[5] = temperature[4];
		temperature[8] = temperature[4];
	}
	
	if (coo.y - 1 >= 0) {
		temperature[0] = temperature_map_device[coo.x - 1][coo.y - 1];
		temperature[1] = temperature_map_device[coo.x][coo.y - 1];
		temperature[2] = temperature_map_device[coo.x + 1][coo.y - 1];
	}
	else {
		temperature[0] = temperature[4];
		temperature[1] = temperature[4];
		temperature[2] = temperature[4];
	}
	
	if (coo.y + 1 < one_side_number) {
		temperature[6] = temperature_map_device[coo.x - 1][coo.y + 1];
		temperature[7] = temperature_map_device[coo.x][coo.y + 1];
		temperature[8] = temperature_map_device[coo.x + 1][coo.y + 1];
	}
	else {
		temperature[6] = temperature[4];
		temperature[7] = temperature[4];
		temperature[8] = temperature[4];
	}

	int sum{};
	for (int i = 0; i < 9; i++) {
		sum += temperature[i];
	}
	//printf("%d\n", sum);

	//temperature_map_device[coo.x][coo.y] = sum / 9;
}

class Terrain
{
private:
	char** terrain_array_host = new char * [one_side_number];
	char** terrain_array_device;
	char* terrain_array_temp[one_side_number];

	char** shadow_map_host = new char* [one_side_number];
	char** shadow_map_device;
	char* shadow_map_temp[one_side_number];
	
	char** temperature_map_host = new char* [one_side_number];
	char** temperature_map_device;
	char* temperature_map_temp[one_side_number];
	
	char** terrain_player_sight_host = new char* [player_sight_size];
	unsigned __int64 init_total_hill_height = 0;
	
	II city_location[5];	//���߿� ũ�� MAXPLAYER�� �����ؾ� ��
	II* city_location_device;

	II* random_array = new II[random_array_size];
	II* random_array_device;
	bool random_array_used = true;
	
	bool log = false;
	
public:
	Terrain()  
	{
		cout << "Generating Terrain Start" << endl;
		
		//�����迭 ���� ������ ����===================================================
		thread t1 = thread(make_random_array, random_array, ref(random_array_used));
		t1.detach();


		//Terrain Memory Assignement===================================================
		clock_t t_1 = clock();
		for (int i = 0; i < one_side_number; i++) {
			terrain_array_host[i] = new char[one_side_number];
			shadow_map_host[i] = new char[one_side_number];
			temperature_map_host[i] = new char[one_side_number];
		}
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				terrain_array_host[i][j] = height_uid(dre);			//��� ���� ���ϰ� �������� ����
				shadow_map_host[i][j] = 0;
				temperature_map_host[i][j] = 30;
			}
		}
		hipMalloc((void**)&terrain_array_device, one_side_number * sizeof(char*));
		hipMalloc((void**)&shadow_map_device, one_side_number * sizeof(char*));
		hipMalloc((void**)&temperature_map_device, one_side_number * sizeof(char*));
		for (int i = 0; i < one_side_number; i++) {
			hipMalloc((void**)&terrain_array_temp[i], one_side_number * sizeof(char));
			hipMalloc((void**)&shadow_map_temp[i], one_side_number * sizeof(char));
			hipMalloc((void**)&temperature_map_temp[i], one_side_number * sizeof(char));
		}
		hipMemcpy(terrain_array_device, terrain_array_temp, one_side_number * sizeof(char*), hipMemcpyHostToDevice);
		hipMemcpy(shadow_map_device, shadow_map_temp, one_side_number * sizeof(char*), hipMemcpyHostToDevice);
		hipMemcpy(temperature_map_device, temperature_map_temp, one_side_number * sizeof(char*), hipMemcpyHostToDevice);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_temp[i], terrain_array_host[i], one_side_number * sizeof(char), hipMemcpyHostToDevice);
			hipMemcpy(shadow_map_temp[i], shadow_map_host[i], one_side_number * sizeof(char), hipMemcpyHostToDevice);
			hipMemcpy(temperature_map_temp[i], temperature_map_host[i], one_side_number * sizeof(char), hipMemcpyHostToDevice);
		}


		//Terrain Memory Assignment For Player's Sight===================================================
		clock_t t_2 = clock();
		for (int i = 0; i < player_sight_size; i++) {
			terrain_player_sight_host[i] = new char[player_sight_size];
		}
		for (int i = 0; i < player_sight_size; i++) {
			for (int j = 0; j < player_sight_size; j++) {
				terrain_player_sight_host[i][j] = 0;
			}
		}


		//Make Hills===================================================
		clock_t t_3 = clock();
		
		init_total_hill_height = add_all();

		cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
		cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
		cout << "Num of Total Blocks: " << init_total_hill_height << endl;
		cout << "Terrain Memory Assignement : " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Terrain Memory Assignment For Player's Sight : " << (double)(t_3 - t_2) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Terrain Completely Generated !" << endl;
		cout << endl;
	}

	~Terrain()
	{
		for (int i = 0; i < one_side_number; i++) {
			delete[] terrain_array_host[i];
			delete[] shadow_map_host[i];
			delete[] temperature_map_host[i];
		}
		for (int i = 0; i < player_sight_size; i++) {
			delete[] terrain_player_sight_host[i];
		}
		delete[] terrain_array_host;
		delete[] shadow_map_host;
		delete[] temperature_map_host;
		delete[] terrain_player_sight_host;
		
		hipFree(terrain_array_temp);
		hipFree(shadow_map_temp);
		hipFree(temperature_map_temp);
		
		hipFree(terrain_array_device);
		hipFree(shadow_map_device);
		hipFree(temperature_map_device);
	}
	
	CC get_highest_lowest()
	{
		clock_t t_0 = clock();
		char highest = terrain_array_host[0][0];
		char lowest = terrain_array_host[0][0];
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				if (terrain_array_host[i][j] > highest) {
					highest = terrain_array_host[i][j];
				}
				else if (terrain_array_host[i][j] < lowest) {
					lowest = terrain_array_host[i][j];
				}
			}
		}
		CC value{ highest, lowest };
		clock_t t_1 = clock();
		cout << "Get Highest : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		return value;
	}

	void make_shadow_map(int sun_angle) {
		clock_t t_0 = clock();
		
		for (int i = 0; i < one_side_number; i++) {
			hipMemset(shadow_map_temp[i], 1, one_side_number * sizeof(char));
		}
		
		if (sun_angle < 0 || sun_angle > 180) {
			cout << "Sun Angle is not valid: " << sun_angle << endl;
			return;
		}
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		make_shadow_map_cuda << <grid, block >> > (terrain_array_device, shadow_map_device, sun_angle);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(shadow_map_host[i], shadow_map_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		
		clock_t t_1 = clock();
		if (log) {
			cout << "Sun Angle: " << sun_angle << endl;
			cout << "Make Shadow Map : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		}
	}

	void make_tempertature_map(int sun_angle)
	{
		clock_t t_0 = clock();
			
		/*for (int i = 0; i < one_side_number; i++) {
			hipMemset(temperature_map_temp[i], 0, one_side_number * sizeof(char));
		}*/
		if (sun_angle < 0 || sun_angle > 180) {
			cout << "Sun Angle is not valid: " << sun_angle << endl;
			return;
		}
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		make_temperature_map_cuda << <grid, block >> > (terrain_array_device, shadow_map_device, temperature_map_device, sun_angle);
		heat_conduction_cuda << <grid, block >> > (temperature_map_device);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(temperature_map_host[i], temperature_map_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		clock_t t_1 = clock();
		if(log)
			cout << "Make Temperature Map : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}

	void terrain_corrosion()
	{
		clock_t t_0 = clock();
		CC hi_low = get_highest_lowest();
		cout << hi_low.x << " " << hi_low.y << endl;
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		for (int height = hi_low.x; height > hi_low.y; height--) {
			terrain_corrosion_cuda << <grid, block >> > (terrain_array_device, height);
		}
		
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		
		clock_t t_1 = clock();
		if (log)
			cout << "Terrain Flatten : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}

	void add_scarce()
	{
		unsigned __int64 scarce_blocks = init_total_hill_height - add_all();
		if (scarce_blocks == 0) {
			return;
		}
		if (log)
			cout<< "scarce_blocks: " << scarce_blocks << endl;

		//==================================================================================
		clock_t t_0 = clock();

		while (random_array_used) {
			//cout << "Waiting for Thread\n";
			Sleep(10);
		}
		
		clock_t t_1 = clock();

		hipMalloc((void**)&random_array_device, random_array_size * sizeof(II));
		hipMemcpy(random_array_device, random_array, random_array_size * sizeof(II), hipMemcpyHostToDevice);
		
		int grid, block;
		if (scarce_blocks <= 1024) {
			grid = 1;
			block = scarce_blocks;
		}
		else {
			grid = scarce_blocks / 1024;
			block = 1024;
		}

		if (scarce_blocks > random_array_size) {
			cout << "FATAL ERROR: scarce_blocks is bigger than random_array_size !!!\n";
			return;
		}
		add_scarce_cuda << <grid, block >> > (terrain_array_device, random_array_device, scarce_blocks);
		hipDeviceSynchronize();
		/*for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}*/
		
		random_array_used = true;
		
		//�޸� ����
		hipFree(random_array_device);
		//==================================================================================

		clock_t t_2 = clock();
		
		/*scarce_blocks = init_total_hill_height - add_all();
		cout << "after_add_blocks: " << scarce_blocks << endl;*/

		if (log) {
			cout << "Waiting Time for Random Thread: " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
			cout << "Add Scarce Cuda: " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " sec" << endl;
		}
	}

	void wind_blow(II wind_direction, int wind_speed)
	{
		clock_t t_0, t_1, t_2, t_3, t_4, t_5;
		
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		
		t_0 = clock();
		
		for (int i = 0; i < wind_speed; i++) {
			//cout << "__________________________" << endl;
			t_1 = clock();

			add_scarce();

			t_2 = clock();
			wind_blow_cuda << <grid, block >> > (terrain_array_device, wind_direction);
			hipDeviceSynchronize();
			/*for (int i = 0; i < one_side_number; i++) {
				hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
			}*/
			
			t_3 = clock();
			if (log)
				cout << "Wind Blow Cuda: " << (double)(t_3 - t_2) / CLOCKS_PER_SEC << " sec" << endl;
			
			except_city_terrain();

			t_4 = clock();
			if (log)
				cout << "=> Once Wind Blow: " << (double)(t_4 - t_1) / CLOCKS_PER_SEC << " sec" << endl;
		}
		t_5 = clock();
		if (log)
			cout << "Total Wind Blow: " << (double)(t_5 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}
	
	unsigned __int64 add_all()
	{
		clock_t t_0 = clock();
		unsigned __int64 all = 0;
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				all += terrain_array_host[i][j];
			}
		}
		clock_t t_1 = clock();
		if (log)
			cout << "Terrain Add All : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		return all;
	}
	
	void except_city_terrain()
	{
		clock_t t_0 = clock();
		city_location_device = new II[5];
		hipMalloc((void**)&city_location_device, 5 * sizeof(II));
		hipMemcpy(city_location_device, city_location, 5 * sizeof(II), hipMemcpyHostToDevice);
		
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		except_city_terrain_cuda << <grid, block >> > (terrain_array_device, city_location_device, 5);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		clock_t t_1 = clock();
		if (log)
			cout << "Except City Terrain: " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}
	
	void show_array(char** terrain_array_host, int size)
	{
		for (int y = 0; y < size; y++) {
			for (int x = 0; x < size; x++) {
				if (terrain_array_host[x][y] > 9) {
					printf("%d_", terrain_array_host[x][y] / 100);
				}
				else if (terrain_array_host[x][y] < 0) {
					printf("%d-", abs(terrain_array_host[x][y]) / 100);
				}
				else
					//printf("%4d", terrain_array_host[x][y]);
					printf("%d ", terrain_array_host[x][y]);

			}
			printf("\n");
			//printf("\n");

		}
	}

	void copy_for_player_map(II player_location)
	{
		clock_t start_t, end_t;
		start_t = clock();
		for (int i = 0; i < player_sight_size; i++) {
			for (int j = 0; j < player_sight_size; j++) {
				if (player_location.x - player_sight_size / 2 + i < 0 || player_location.x - player_sight_size / 2 + i >= one_side_number || player_location.y - player_sight_size / 2 + j < 0 || player_location.y - player_sight_size / 2 + j >= one_side_number)
					terrain_player_sight_host[i][j] = 0;
				else
					terrain_player_sight_host[i][j] = terrain_array_host[player_location.x - player_sight_size / 2 + i][player_location.y - player_sight_size / 2 + j];
			}
		}
		end_t = clock();
		if (log)
			cout << "copy_for_player_map : " << double(end_t - start_t) / CLOCKS_PER_SEC << endl;
	}
	
	void get_device_info()
	{
		hipDeviceProp_t  prop;

		int count;
		hipGetDeviceCount(&count);

		for (int i = 0; i < count; i++) {
			hipGetDeviceProperties(&prop, i);
			printf("   --- General Information for device %d ---\n", i);
			printf("Name:  %s\n", prop.name);
			printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
			printf("Clock rate:  %d\n", prop.clockRate);
			printf("Device copy overlap:  ");
			if (prop.deviceOverlap)
				printf("Enabled\n");
			else
				printf("Disabled\n");
			printf("Kernel execution timeout :  ");
			if (prop.kernelExecTimeoutEnabled)
				printf("Enabled\n");
			else
				printf("Disabled\n");
			printf("\n");

			printf("   --- Memory Information for device %d ---\n", i);
			printf("Total global mem:  %ld\n", prop.totalGlobalMem);
			printf("Total constant Mem:  %ld\n", prop.totalConstMem);
			printf("Max mem pitch:  %ld\n", prop.memPitch);
			printf("Texture Alignment:  %ld\n", prop.textureAlignment);
			printf("\n");

			printf("   --- MP Information for device %d ---\n", i);
			printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
			printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
			printf("Registers per mp:  %d\n", prop.regsPerBlock);
			printf("Threads in warp:  %d\n", prop.warpSize);
			printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
			printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
			printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
			printf("\n");
		}
	}
	
	template <typename T>
	int delete_array(T* array, int i, int size)
	{
		for (int j = i; j < size; j++) {
			array[j] = array[j + 1];
		}
		size -= 1;
		return size;
	}

	int make_hill_location(HI* hill_location_host)
	{
		int num_of_hills = number_of_hills_uid(dre);
		cout << "expected num of hills: " << num_of_hills << endl;

		for (int i = 0; i < num_of_hills; i++) {
			hill_location_host[i].x = terrain_distance(dre);
			hill_location_host[i].y = terrain_distance(dre);
			hill_location_host[i].radius = hill_size_uid(dre);
			hill_location_host[i].height = height_uid(dre);
		}
		//sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
		for (int a = 0; a < num_of_hills; a++) {
			for (int b = 0; b < num_of_hills; b++) {
				if (a != b) {
					if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
							num_of_hills = delete_array(hill_location_host, b, num_of_hills);
							b--;
						}
					}
				}
			}
		}
		/*for (int i = 0; i < num_of_hills; i++) {
			if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
				num_of_hills = delete_array (hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
		}*/
		cout << "real num of hills: " << num_of_hills << endl;
		return num_of_hills;
	}

	void make_new_hills(HI* hill_location_host, int& num_of_hills, int origin_num_of_hills, FF wind_direction, int wind_speed)
	{
		cout << "Wind info: " << wind_direction.x << " " << wind_direction.y << endl;
		if (wind_speed) {
			hill_location_host[num_of_hills].radius = hill_size_uid(dre);
			hill_location_host[num_of_hills].height = height_uid(dre);

			hill_location_host[num_of_hills].x = terrain_distance(dre);
			hill_location_host[num_of_hills].y = terrain_distance(dre);

			while (1) {
				hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
				hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
				if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
					break;
				}
				if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
					break;
				}
				if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
					break;
				}
				if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
					break;
				}
			}

			int collide_iter{};
			for (int a = 0; a < num_of_hills; a++) {
				if (collide_iter > 10) {	//���ѷ��� ���� ���ɼ����� ���� Ƚ�� ����
					return;
				}
				//cout << "���: "<<a << " " << hill_location_host[a].x << " " << hill_location_host[a].y << " " << hill_location_host[a].radius << " " << hill_location_host[a].height << endl;
				//cout << " ��: "<< num_of_hills<< " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills ].y << " " << hill_location_host[num_of_hills ].radius << " " << hill_location_host[num_of_hills].height << endl << endl;

				if (pow(hill_location_host[a].x - hill_location_host[num_of_hills].x, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {
					if (pow(hill_location_host[a].y - hill_location_host[num_of_hills].y, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {

						hill_location_host[num_of_hills].radius = hill_size_uid(dre);
						hill_location_host[num_of_hills].height = height_uid(dre);

						hill_location_host[num_of_hills].x = terrain_distance(dre);
						hill_location_host[num_of_hills].y = terrain_distance(dre);

						while (1) {
							hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
							hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
							if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
								break;
							}
							if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
								break;
							}
							if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
								break;
							}
							if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
								break;
							}
						}
						cout << "�浹�� ���� �ٲ�" << endl;
						a = -1;
						collide_iter++;
					}
				}
			}
			cout << " ����: " << num_of_hills << " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills].y << " " << hill_location_host[num_of_hills].radius << " " << hill_location_host[num_of_hills].height << endl << endl;
			num_of_hills++;
		}
	}

	void move_terrain(HI* hill_location_host, int& num_of_hills, FF wind_direction, int wind_speed)
	{
		if (wind_speed) {
			int wind_move_x = wind_speed * wind_direction.x;
			int wind_move_y = wind_speed * wind_direction.y;
			cout << "Wind == " << "X: " << wind_move_x << " " << "Y: " << wind_move_y << endl;
			for (int i = 0; i < num_of_hills; i++) {
				hill_location_host[i].x += wind_move_x;
				hill_location_host[i].y += wind_move_y;

				if (hill_location_host[i].x - hill_location_host[i].radius > one_side_number) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].x + hill_location_host[i].radius < 0) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].y - hill_location_host[i].radius > one_side_number) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].y + hill_location_host[i].radius < 0) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
			}
			/*for (int i = 0; i < num_of_hills; i++) {
				cout << hill_location_host[i].x << " " << hill_location_host[i].y << endl;
			}
			cout << endl;*/
		}
	}

	void wind_decide(int& wind_speed, int& wind_angle)
	{
		//wind speed 0-50
		//wind angle 0-360
		wind_speed = 50;// wind_speed_uid(dre);
		//wind_angle = 90; // wind_angle_uid(dre);
		wind_angle += 10;
		cout << wind_speed << " " << wind_angle << endl;
		//ǳ���� �������� �ѹ� ������Ʈ �� ������, ǳ���� �������� �ѹ� ������Ʈ �� ������ ȸ�Ǹ� ���� ��������
	}
	
	char** get_map() {
		return terrain_array_host;
	}

	char** get_shadow_map() {
		return shadow_map_host;
	}
	
	char** get_temperature_map()
	{
		return temperature_map_host;
	}

	char** get_player_sight_map() {
		return terrain_player_sight_host;
	}

	void set_city_location(TF location, int iter) {
		if (iter > 4) {
			cout << "Error: set_city_location, Bigger than array size" << endl;
		}
		city_location[iter].x = location.x;
		city_location[iter].y = location.y;
		cout << "city_location[" << iter << "] = " << city_location[iter].x << " " << city_location[iter].y << endl;
	}

	void log_on()
	{
		log = true;
	}

};
