#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <random>
#include <windows.h>
#include <time.h>
#include <math.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include ""
#include "global.h"
#define PI 3.1415926
using namespace std;

const int one_side_number = 39936;	//39936
const int player_sight_size = 64;	//1024 ������ �ȵ�
const int random_array_size = 150000000;// 150000000;

const int max_height = 8;
const int base_floor = 1;

typedef struct two_int {
	int x;
	int y;
} II;
typedef struct two_char {
	int x;
	int y;
} CC;
typedef struct two_float {
	float x;
	float y;
} FF;
typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;
typedef struct compare_block {
	II middle;
	II up;
	II down;
	II left;
	II right;
}CB;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>terrain_distance(0, one_side_number - 1);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 10);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(4, max_height);

uniform_int_distribution <int>wind_speed_uid(0, 50);
uniform_int_distribution <int>wind_angle_uid(0, 360);

void make_random_array(II* random_array, bool& random_array_used)
{
	while (1) {
		if (random_array_used) {
			clock_t t_0 = clock();
			for (int i = 0; i < random_array_size; i++) {
				random_array[i].x = terrain_distance(dre);
				random_array[i].y = terrain_distance(dre);
			}
			random_array_used = false;
			clock_t t_1 = clock();
			cout << "_Thread_ Random for Scarce: " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec\n";
		}
		else {
			Sleep(10);
		}
	}
}

__global__
void make_hills_cuda(char** terrain_array_device, HI* hill_location_device, int num_of_hills)
{
	II terrain;
	terrain.x = blockIdx.x * blockDim.x + threadIdx.x;
	terrain.y = blockIdx.y * blockDim.y + threadIdx.y;
	HI hill;
	int distance;

	if (terrain.x % 5 == 0 && terrain.y % 5 == 0) {
		terrain_array_device[terrain.x][terrain.y] += 4;
	}


	//terrain_array_device[terrain.x][terrain.y] = base_floor;
	//for (int i = 0; i < num_of_hills; i++) {
	//	//���� ��� ä���
	//	hill.x = hill_location_device[i].x;
	//	hill.y = hill_location_device[i].y;
	//	hill.radius = hill_location_device[i].radius;
	//	hill.height = hill_location_device[i].height;
	//	distance = sqrt(pow(terrain.y - hill.y, 2) + pow(terrain.x - hill.x, 2));

	//	if (distance <= hill.radius) {
	//		terrain_array_device[terrain.x][terrain.y] += (hill.height) * (hill.radius - distance) / hill.radius;
	//		return;
	//	}
	//}
}

__global__
void terrain_corrosion_cuda(char** terrain_array_device, int height)
{
	//�ٶ��� �Ҹ� �������� �̵��ϰԲ� �����ؾ� ��
	//�ƴϸ� �ٶ����� ���� �Լ� ���� ���� �ߺ����� ���� �� ��� Ȯ��
	const int block_num = 9;
	II terrain[block_num];
	terrain[0].x = blockIdx.x * blockDim.x + threadIdx.x;	//middle
	terrain[0].y = blockIdx.y * blockDim.y + threadIdx.y;

	if (terrain_array_device[terrain[0].x][terrain[0].y] != height) {	//���ڷ� ���� ���̰� �ƴϸ� ����
		return;
	}
	if (terrain_array_device[terrain[0].x][terrain[0].y] <= base_floor) {	//base floor ���� ������ ���̻� ���߸� �ȵ�
		return;
	}
	
	terrain[1] = { terrain[0].x, terrain[0].y - 1 };		//up
	terrain[2] = { terrain[0].x, terrain[0].y + 1 };		//down
	terrain[3] = { terrain[0].x - 1, terrain[0].y };		//left
	terrain[4] = { terrain[0].x + 1, terrain[0].y };		//right
	terrain[5] = { terrain[0].x - 1, terrain[0].y - 1 };	//up left
	terrain[6] = { terrain[0].x - 1, terrain[0].y + 1 };	//down left
	terrain[7] = { terrain[0].x + 1, terrain[0].y - 1 };	//up right
	terrain[8] = { terrain[0].x + 1, terrain[0].y + 1 };	//down right

	//�����¿� ����
	for (int i = 0; i < block_num; i++) {
		if (terrain[i].x < 0) {
			terrain[i].x += one_side_number;
		}
		if (terrain[i].x >= one_side_number) {
			terrain[i].x -= one_side_number;
		}
		if (terrain[i].y < 0) {
			terrain[i].y += one_side_number;
		}
		if (terrain[i].y >= one_side_number) {
			terrain[i].y -= one_side_number;
		}
	}

	int height_difference = 0;
	for (int i = 1; i < block_num; i++) {	//�ֺ� ������ ���� ���̸� ����
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] > height_difference) {
			height_difference = terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y];
		}
	}

	if (height_difference <= 0) {	//�ֺ��� �� ���� ���� ������ ����
		return;
	}

	int num_of_lowest{};
	for (int i = 1; i < block_num; i++) {
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
			num_of_lowest++;
		}
	}

	if (num_of_lowest == 1) {	//���� ���� ���̰� �ϳ��� �ű�� �̵�
		for (int i = 1; i < block_num; i++) {
			if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
				terrain_array_device[terrain[i].x][terrain[i].y]++;
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
		}
	}
	if (num_of_lowest == 8 && height_difference == 1) {	//�ֺ��� ��� ���̰� ���� ���� ���̰� 1�̸� ����
		return;
	}
	
	char thread_seed = (terrain[0].x + terrain[0].y + abs(terrain[0].x + terrain[0].y * height)) % num_of_lowest;	//random seed made by myself
	int iter = 0;
	for (int i = 0; i < block_num; i++) {
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {	//���� ���� ���̰� �������� �������� �ϳ� ����
			if (thread_seed == iter) {	//�������� ���õ� ������ �̵�
				terrain_array_device[terrain[i].x][terrain[i].y]++;
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
			iter++;
		}
	}
}

__global__
void wind_blow_cuda(char** terrain_array_device, II wind_direction)
{
	//wind_direction�� x, y�� �ϳ��� ������ 0�̿��� ��
	II terrain[5];
	terrain[0].x = blockIdx.y * blockDim.y + threadIdx.y;	//����ϰԵ� x,y�°� �ϸ� ���� ���� ��ȭ�ҽ� Ư�� ���̰� ����. �̷��� �ϸ� �Ȼ���. �̷��� �ٲ㵵 �������� ���� ��
	terrain[0].y = blockIdx.x * blockDim.x + threadIdx.x;	//middle

	terrain[1].x = terrain[0].x + wind_direction.x;			//forward
	terrain[1].y = terrain[0].y + wind_direction.y;

	terrain[2].x = terrain[1].x + wind_direction.y;			//forward left
	terrain[2].y = terrain[1].y - wind_direction.x;
	
	terrain[3].x = terrain[1].x - wind_direction.y;			//forward right
	terrain[3].y = terrain[1].y + wind_direction.x;

	terrain[4].x = terrain[0].x - wind_direction.x;			//back
	terrain[4].y = terrain[0].y - wind_direction.y;
	
	//���� �����¿� ����
	for (int i = 0; i < 5; i++) 
	{
		if (terrain[i].x < 0 ) {
			terrain[i].x += one_side_number;
		}
		if (terrain[i].x >= one_side_number) {
			terrain[i].x -= one_side_number;
		}
		if (terrain[i].y < 0) {
			terrain[i].y += one_side_number;
		}
		if (terrain[i].y >= one_side_number) {
			terrain[i].y -= one_side_number;
		}
	}

	if (terrain_array_device[terrain[0].x][terrain[0].y] <= base_floor) 
	{	//base floor ���� ������ ���̻� ���߸� �ȵ�
		return;
	}
	//�̰Ŷ����� ������ ������
	if (terrain_array_device[terrain[0].x][terrain[0].y] > base_floor && terrain_array_device[terrain[0].x][terrain[0].y] == terrain_array_device[terrain[4].x][terrain[4].y] && terrain_array_device[terrain[0].x][terrain[0].y] == terrain_array_device[terrain[1].x][terrain[1].y]) {	//��� �þ�� �������� ����
		
	}
	else if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[4].x][terrain[4].y] <= 0) 
	{	//�Ĺ� ���� �� ������ ���ų� ���� ��� �ٶ��� ������ ���� ����
		return;
	}

	int height_difference = -1;
	for (int i = 1; i < 4; i++) 
	{
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y]  > height_difference) 
		{
			height_difference = terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y];	//i���� 0���� ���� ����
		}
	}
	if (height_difference < 0) {	//���� 3�� ������ ���̰� ��� �� ������ ���� ��� ���� �ȹ���
		return;
	}

	int num_of_lowest{};
	for (int i = 1; i < 4; i++) 
	{
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) 
		{	//���� ���� ���� �����
			num_of_lowest++;
		}
	}

	if (num_of_lowest == 0)
		return; 
	
	//�������̰� �ϳ��� ��� �׸��� ��
	if (num_of_lowest == 1) 
	{	
		for (int i = 1; i < 4; i++) 
		{
			if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) 
			{
				--terrain_array_device[terrain[0].x][terrain[0].y];
				++terrain_array_device[terrain[i].x][terrain[i].y];
				return;
			}
		}
	}
	
	//�������̰� ������ �� ��, ���� ���� ���� �����̸� �������� ����
	if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[1].x][terrain[1].y] == height_difference) 
	{		
		--terrain_array_device[terrain[0].x][terrain[0].y];
		++terrain_array_device[terrain[1].x][terrain[1].y];
		return;
	}

	int radom_seed = (terrain[0].x + terrain[0].y) % num_of_lowest + 2;
	//�� �� ���� �ϳ��� ���� �̵�
	--terrain_array_device[terrain[0].x][terrain[0].y];
	++terrain_array_device[terrain[radom_seed].x][terrain[radom_seed].y];
	return;
}

__global__
void add_scarce_cuda(char** terrain_array_device, II* random_seed_device, int size) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	II my_seed;
	my_seed.x = random_seed_device[id].x;
	my_seed.y = random_seed_device[id].y;

	terrain_array_device[my_seed.x][my_seed.y]++;
}

__global__
void player_terrain_update_cuda(char** terrain_player_sight_device, HI* hill_location_device, int num_of_hills, II player_location, FF wind_direction, int wind_speed)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int terrain_x = player_location.x + x;
	int terrain_y = player_location.y + y;

	if (terrain_x >= 0 && terrain_x <= one_side_number && terrain_y >= 0 && terrain_y <= one_side_number) {
		terrain_player_sight_device[x][y] = base_floor;
		for (int i = 0; i < num_of_hills; i++) {
			
			//���� ��� ä���
			int hill_location_x = hill_location_device[i].x;
			int hill_location_y = hill_location_device[i].y;
			int radius = hill_location_device[i].radius;
			int height = hill_location_device[i].height;
			int distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

			if (distance <= radius) {
				terrain_player_sight_device[x][y] += (height) * (radius - distance) / radius;

				//��� ���
				hill_location_x = hill_location_device[i].x - radius * wind_direction.x * (100 - wind_speed) / 50;
				hill_location_y = hill_location_device[i].y - radius * wind_direction.y * (100 - wind_speed) / 50;
				distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

				if (distance <= radius) {
					terrain_player_sight_device[x][y] -= height * (radius - distance) / radius + base_floor;
					if (terrain_player_sight_device[x][y] > max_height) {
						terrain_player_sight_device[x][y] = max_height;
					}
					else if (terrain_player_sight_device[x][y] < base_floor) {
						terrain_player_sight_device[x][y] = base_floor;
					}
				}
			}
		}
	}
	else {
		//�� ���� ��� 0
		terrain_player_sight_device[x][y] = 0;
	}
}

__global__
void except_city_terrain_cuda(char** terrain_array_device, II* city_location_device, int num_of_city)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int outer_radius = 40;
	int inner_radius = outer_radius - 10;
	II distance;

	for (int i = 0; i < num_of_city; i++) {
		if (city_location_device[i].x == 0 && city_location_device[i].y == 0) 
			continue;
		if (abs(x - city_location_device[i].x) <= inner_radius && abs(y - city_location_device[i].y) <= inner_radius) {
			terrain_array_device[x][y] = 1;
			return;
		}
		
		distance.x = abs(x - city_location_device[i].x);
		distance.y = abs(y - city_location_device[i].y);
		//printf("%d, %d\n", distance.x, distance.y);
		if (distance.x > outer_radius || distance.y > outer_radius)
			continue;
		
		if (distance.x == distance.y && distance.x == 0) 
			terrain_array_device[x][y] -= outer_radius;
		else if (distance.x >= distance.y) 
			terrain_array_device[x][y] += distance.x - outer_radius;
		else if (distance.x < distance.y) 
			terrain_array_device[x][y] += distance.y - outer_radius;
		
		if (terrain_array_device[x][y] < 1) 
			terrain_array_device[x][y] = 1;
	}
}

class Terrain
{
private:
	char** terrain_array_host = new char* [one_side_number];
	char** terrain_array_device;
	char* terrain_array_temp[one_side_number];

	char** terrain_player_sight_host = new char* [player_sight_size];
	unsigned __int64 init_total_hill_height = 0;
	
	II city_location[5];	//���߿� ũ�� MAXPLAYER�� �����ؾ� ��
	II* city_location_device;

	II* random_array = new II[random_array_size];
	II* random_array_device;
	bool random_array_used = true;
	
	
public:
	Terrain()  
	{
		cout << "Generating Terrain Start" << endl;
		
		//�����迭 ���� ������ ����
		thread t1 = thread(make_random_array, random_array, ref(random_array_used));
		t1.detach();
		
		//Make Random Hills Information===================================================
		//clock_t t_0 = clock();

		//HI* hill_location_host;
		//HI* hill_location_device;
		//hill_location_host = new HI[4000];
		//hill_location_device;
		//hipMalloc((void**)&hill_location_device, 4000 * sizeof(HI));
		//int num_of_hills = make_hill_location(hill_location_host);
		//hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device
		//printf("Random Hill Info Complete\n");
		//for (int i = 0; i < num_of_hills; i++) {
		//	cout << hill_location_host[i].x << ", " << hill_location_host[i].y << ", " << hill_location_host[i].height << ", " << hill_location_host[i].radius << endl;
		//}


		//Terrain Memory Assignement===================================================
		clock_t t_1 = clock();
		for (int i = 0; i < one_side_number; i++) {
			terrain_array_host[i] = new char[one_side_number];
		}
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				terrain_array_host[i][j] = height_uid(dre);			//��� ���� ���ϰ� �������� ����
			}
		}
		hipMalloc((void**)&terrain_array_device, one_side_number * sizeof(char*));
		for (int i = 0; i < one_side_number; i++) {
			hipMalloc((void**)&terrain_array_temp[i], one_side_number * sizeof(char));
		}
		hipMemcpy(terrain_array_device, terrain_array_temp, one_side_number * sizeof(char*), hipMemcpyHostToDevice);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_temp[i], terrain_array_host[i], one_side_number * sizeof(char), hipMemcpyHostToDevice);
		}


		//Terrain Memory Assignment For Player's Sight===================================================
		clock_t t_2 = clock();
		for (int i = 0; i < player_sight_size; i++) {
			terrain_player_sight_host[i] = new char[player_sight_size];
		}
		for (int i = 0; i < player_sight_size; i++) {
			for (int j = 0; j < player_sight_size; j++) {
				terrain_player_sight_host[i][j] = 0;
			}
		}


		//Make Hills===================================================
		clock_t t_3 = clock();
		/*dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		make_hills_cuda << <grid, block >> > (terrain_array_device, hill_location_device, num_of_hills);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}*/
		
		clock_t  t_4 = clock();
		init_total_hill_height = add_all();

		cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
		cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
		cout << "Num of Total Blocks: " << init_total_hill_height << endl;
		//cout << "Make Random Hills Information : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Terrain Memory Assignement : " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Terrain Memory Assignment For Player's Sight : " << (double)(t_3 - t_2) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Make Hills (GPU) : " << (double)(t_4 - t_3) / CLOCKS_PER_SEC << " sec" << endl;
		//cout << "Make Hills (CPU) : " << (double)(t_6 - t_5) / CLOCKS_PER_SEC << " sec" << endl;

		cout << "Terrain Completely Generated !" << endl;
		cout << endl;
	}

	~Terrain()
	{
		for (int i = 0; i < one_side_number; i++) {
			delete[] terrain_array_host[i];
		}
		for (int i = 0; i < player_sight_size; i++) {
			delete[] terrain_player_sight_host[i];
		}
		delete[] terrain_array_host;
		delete[] terrain_player_sight_host;
		hipFree(terrain_array_temp);
		hipFree(terrain_array_device);
	}
	
	void make_hills_cpu(char** terrain_array_host, HI* hill_location_host, int hill_number)
	{
		II terrain;
		HI hill;
		int distance;
		for (int i = 0; i < one_side_number; i++) {
			terrain.y = i;
			for (int j = 0; j < one_side_number; j++) {
				terrain.x = j;
				
				//printf("%d %d %d\n", terrain.x, terrain.y, i);
				//terrain_array_device[terrain.x][terrain.y] = base_floor;
				for (int k = 0; k < hill_number; k++) {
					//���� ��� ä���
					hill.x = hill_location_host[k].x;
					hill.y = hill_location_host[k].y;
					hill.radius = hill_location_host[k].radius;
					hill.height = hill_location_host[k].height;
					distance = sqrt(pow(terrain.y - hill.y, 2) + pow(terrain.x - hill.x, 2));
					//cout << distance << endl;

					if (distance <= hill.radius) {
						//cout << hill.x << " " << hill.y << endl;

						terrain_array_host[terrain.x][terrain.y] += (hill.height) * (hill.radius - distance) / hill.radius;
					}
				}
			}
		}
	}
	
	CC get_highest_lowest()
	{
		clock_t t_0 = clock();
		char highest = terrain_array_host[0][0];
		char lowest = terrain_array_host[0][0];
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				if (terrain_array_host[i][j] > highest) {
					highest = terrain_array_host[i][j];
				}
				else if (terrain_array_host[i][j] < lowest) {
					lowest = terrain_array_host[i][j];
				}
			}
		}
		CC value{ highest, lowest };
		clock_t t_1 = clock();
		cout << "Get Highest : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		return value;
	}

	void terrain_corrosion()
	{
		clock_t t_0 = clock();
		CC hi_low = get_highest_lowest();
		cout << hi_low.x << " " << hi_low.y << endl;
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		for (int height = hi_low.x; height > hi_low.y; height--) {
			terrain_corrosion_cuda << <grid, block >> > (terrain_array_device, height);
		}
		
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		
		clock_t t_1 = clock();
		cout << "Terrain Flatten : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}

	void add_scarce()
	{
		unsigned __int64 scarce_blocks = init_total_hill_height - add_all();
		if (scarce_blocks == 0) {
			return;
		}
		cout<< "scarce_blocks: " << scarce_blocks << endl;

		//==================================================================================
		clock_t t_0 = clock();

		while (random_array_used) {
			cout << "Waiting for Thread\n";
			Sleep(10);
		}
		
		clock_t t_1 = clock();

		hipMalloc((void**)&random_array_device, random_array_size * sizeof(II));
		hipMemcpy(random_array_device, random_array, random_array_size * sizeof(II), hipMemcpyHostToDevice);
		
		int grid, block;
		if (scarce_blocks <= 1024) {
			grid = 1;
			block = scarce_blocks;
		}
		else {
			grid = scarce_blocks / 1024;
			block = 1024;
		}

		if (scarce_blocks > random_array_size) {
			cout << "FATAL ERROR: scarce_blocks is bigger than random_array_size !!!\n";
			return;
		}
		add_scarce_cuda << <grid, block >> > (terrain_array_device, random_array_device, scarce_blocks);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		random_array_used = true;
		
		//�޸� ����
		hipFree(random_array_device);
		//==================================================================================

		//int grid, block;
		//if (scarce_blocks <= 1024) {
		//	grid = 1;
		//	block = scarce_blocks;
		//}
		//else {
		//	grid = scarce_blocks / 1024;
		//	block = 1024;
		//}
		////cout << "Grid * Block: " << grid * block << endl;
		//add_scarce_cuda << <grid, block >> > (terrain_array_device, random_seed_device, scarce_blocks);
		//for (int i = 0; i < one_side_number; i++) {
		//	hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		//}
		////�޸� ����
		//delete[] random_seed;
		//hipFree(random_seed_device);
		
		clock_t t_2 = clock();
		
		/*scarce_blocks = init_total_hill_height - add_all();
		cout << "after_add_blocks: " << scarce_blocks << endl;*/

		cout << "Waiting Time for Random Thread: " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Add Scarce Cuda: " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " sec" << endl;
	}

	void wind_blow(II wind_direction, int wind_speed)
	{
		/*FF wind_direction = { cos(wind_angle * PI / 180), sin(wind_angle * PI / 180) };
		if (abs(wind_direction.x) < FLT_EPSILON) {
			wind_direction.x = 0;
		}
		if (abs(wind_direction.y) < FLT_EPSILON) {
			wind_direction.y = 0;
		}*/
		clock_t t_0, t_1, t_2, t_3;
		
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		
		t_0 = clock();

		for (int i = 0; i < wind_speed; i++) {
			cout << "__________________________" << endl;
			add_scarce();

			t_1 = clock();

			wind_blow_cuda << <grid, block >> > (terrain_array_device, wind_direction);
			for (int i = 0; i < one_side_number; i++) {
				hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
			}
			t_2 = clock();
			cout << "Only Wind Blow Cuda: " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " sec" << endl;
		}

		t_3 = clock();
		cout << "[Total Wind Blow: " << (double)(t_3 - t_0) / CLOCKS_PER_SEC << " sec]" << endl;
	}
	
	unsigned __int64 add_all()
	{
		clock_t t_0 = clock();
		unsigned __int64 all = 0;
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				all += terrain_array_host[i][j];
			}
		}
		clock_t t_1 = clock();
		cout << "Terrain Add All : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		return all;
	}
	
	void except_city_terrain()
	{
		clock_t t_0 = clock();
		city_location_device = new II[5];
		hipMalloc((void**)&city_location_device, 5 * sizeof(II));
		hipMemcpy(city_location_device, city_location, 5 * sizeof(II), hipMemcpyHostToDevice);
		
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		except_city_terrain_cuda << <grid, block >> > (terrain_array_device, city_location_device, 5);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		clock_t t_1 = clock();
		cout << "Except City Terrain: " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}
	
	void show_array(char** terrain_array_host, int size)
	{
		for (int y = 0; y < size; y++) {
			for (int x = 0; x < size; x++) {
				if (terrain_array_host[x][y] > 9) {
					printf("H ");
				}
				else
					printf("%d ", terrain_array_host[x][y]);
			}
			printf("\n");
		}
	}

	void copy_for_player_map(II player_location)
	{
		clock_t start_t, end_t;
		start_t = clock();
		for (int i = 0; i < player_sight_size; i++) {
			for (int j = 0; j < player_sight_size; j++) {
				if (player_location.x - player_sight_size / 2 + i < 0 || player_location.x - player_sight_size / 2 + i >= one_side_number || player_location.y - player_sight_size / 2 + j < 0 || player_location.y - player_sight_size / 2 + j >= one_side_number)
					terrain_player_sight_host[i][j] = 0;
				else
					terrain_player_sight_host[i][j] = terrain_array_host[player_location.x - player_sight_size / 2 + i][player_location.y - player_sight_size / 2 + j];
			}
		}
		end_t = clock();
		cout << "copy_for_player_map : " << double(end_t - start_t) / CLOCKS_PER_SEC << endl;
	}
	
	void get_device_info()
	{
		hipDeviceProp_t  prop;

		int count;
		hipGetDeviceCount(&count);

		for (int i = 0; i < count; i++) {
			hipGetDeviceProperties(&prop, i);
			printf("   --- General Information for device %d ---\n", i);
			printf("Name:  %s\n", prop.name);
			printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
			printf("Clock rate:  %d\n", prop.clockRate);
			printf("Device copy overlap:  ");
			if (prop.deviceOverlap)
				printf("Enabled\n");
			else
				printf("Disabled\n");
			printf("Kernel execution timeout :  ");
			if (prop.kernelExecTimeoutEnabled)
				printf("Enabled\n");
			else
				printf("Disabled\n");
			printf("\n");

			printf("   --- Memory Information for device %d ---\n", i);
			printf("Total global mem:  %ld\n", prop.totalGlobalMem);
			printf("Total constant Mem:  %ld\n", prop.totalConstMem);
			printf("Max mem pitch:  %ld\n", prop.memPitch);
			printf("Texture Alignment:  %ld\n", prop.textureAlignment);
			printf("\n");

			printf("   --- MP Information for device %d ---\n", i);
			printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
			printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
			printf("Registers per mp:  %d\n", prop.regsPerBlock);
			printf("Threads in warp:  %d\n", prop.warpSize);
			printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
			printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
			printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
			printf("\n");
		}
	}
	
	template <typename T>
	int delete_array(T* array, int i, int size)
	{
		for (int j = i; j < size; j++) {
			array[j] = array[j + 1];
		}
		size -= 1;
		return size;
	}

	int make_hill_location(HI* hill_location_host)
	{
		int num_of_hills = number_of_hills_uid(dre);
		cout << "expected num of hills: " << num_of_hills << endl;

		for (int i = 0; i < num_of_hills; i++) {
			hill_location_host[i].x = terrain_distance(dre);
			hill_location_host[i].y = terrain_distance(dre);
			hill_location_host[i].radius = hill_size_uid(dre);
			hill_location_host[i].height = height_uid(dre);
		}
		//sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
		for (int a = 0; a < num_of_hills; a++) {
			for (int b = 0; b < num_of_hills; b++) {
				if (a != b) {
					if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
							num_of_hills = delete_array(hill_location_host, b, num_of_hills);
							b--;
						}
					}
				}
			}
		}
		/*for (int i = 0; i < num_of_hills; i++) {
			if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
				num_of_hills = delete_array (hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
		}*/
		cout << "real num of hills: " << num_of_hills << endl;
		return num_of_hills;
	}

	void make_new_hills(HI* hill_location_host, int& num_of_hills, int origin_num_of_hills, FF wind_direction, int wind_speed)
	{
		cout << "Wind info: " << wind_direction.x << " " << wind_direction.y << endl;
		if (wind_speed) {
			hill_location_host[num_of_hills].radius = hill_size_uid(dre);
			hill_location_host[num_of_hills].height = height_uid(dre);

			hill_location_host[num_of_hills].x = terrain_distance(dre);
			hill_location_host[num_of_hills].y = terrain_distance(dre);

			while (1) {
				hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
				hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
				if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
					break;
				}
				if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
					break;
				}
				if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
					break;
				}
				if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
					break;
				}
			}

			int collide_iter{};
			for (int a = 0; a < num_of_hills; a++) {
				if (collide_iter > 10) {	//���ѷ��� ���� ���ɼ����� ���� Ƚ�� ����
					return;
				}
				//cout << "���: "<<a << " " << hill_location_host[a].x << " " << hill_location_host[a].y << " " << hill_location_host[a].radius << " " << hill_location_host[a].height << endl;
				//cout << " ��: "<< num_of_hills<< " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills ].y << " " << hill_location_host[num_of_hills ].radius << " " << hill_location_host[num_of_hills].height << endl << endl;

				if (pow(hill_location_host[a].x - hill_location_host[num_of_hills].x, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {
					if (pow(hill_location_host[a].y - hill_location_host[num_of_hills].y, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {

						hill_location_host[num_of_hills].radius = hill_size_uid(dre);
						hill_location_host[num_of_hills].height = height_uid(dre);

						hill_location_host[num_of_hills].x = terrain_distance(dre);
						hill_location_host[num_of_hills].y = terrain_distance(dre);

						while (1) {
							hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
							hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
							if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
								break;
							}
							if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
								break;
							}
							if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
								break;
							}
							if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
								break;
							}
						}
						cout << "�浹�� ���� �ٲ�" << endl;
						a = -1;
						collide_iter++;
					}
				}
			}
			cout << " ����: " << num_of_hills << " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills].y << " " << hill_location_host[num_of_hills].radius << " " << hill_location_host[num_of_hills].height << endl << endl;
			num_of_hills++;
		}
	}

	void move_terrain(HI* hill_location_host, int& num_of_hills, FF wind_direction, int wind_speed)
	{
		if (wind_speed) {
			int wind_move_x = wind_speed * wind_direction.x;
			int wind_move_y = wind_speed * wind_direction.y;
			cout << "Wind == " << "X: " << wind_move_x << " " << "Y: " << wind_move_y << endl;
			for (int i = 0; i < num_of_hills; i++) {
				hill_location_host[i].x += wind_move_x;
				hill_location_host[i].y += wind_move_y;

				if (hill_location_host[i].x - hill_location_host[i].radius > one_side_number) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].x + hill_location_host[i].radius < 0) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].y - hill_location_host[i].radius > one_side_number) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].y + hill_location_host[i].radius < 0) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
			}
			/*for (int i = 0; i < num_of_hills; i++) {
				cout << hill_location_host[i].x << " " << hill_location_host[i].y << endl;
			}
			cout << endl;*/
		}
	}

	void wind_decide(int& wind_speed, int& wind_angle)
	{
		//wind speed 0-50
		//wind angle 0-360
		wind_speed = 50;// wind_speed_uid(dre);
		//wind_angle = 90; // wind_angle_uid(dre);
		wind_angle += 10;
		cout << wind_speed << " " << wind_angle << endl;
		//ǳ���� �������� �ѹ� ������Ʈ �� ������, ǳ���� �������� �ѹ� ������Ʈ �� ������ ȸ�Ǹ� ���� ��������
	}
	
	char** get_map() {
		return terrain_array_host;
	}

	char** get_player_sight_map() {
		return terrain_player_sight_host;
	}

	void set_city_location(TF location, int iter) {
		if (iter > 4) {
			cout << "Error: set_city_location, Bigger than array size" << endl;
		}
		city_location[iter].x = location.x;
		city_location[iter].y = location.y;
		cout << "city_location[" << iter << "] = " << city_location[iter].x << " " << city_location[iter].y << endl;
	}
};
