#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <random>
#include <windows.h>
#include <time.h>
#include <math.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include ""
#define PI 3.1415926
using namespace std;

const int one_side_number = 96;	//39936
const int player_sight_size = 1024;	//1024 ������ �ȵ�

const int max_height = 8;
const int base_floor = 1;

typedef struct two_int {
	int x;
	int y;
} II;

typedef struct two_char {
	int x;
	int y;
} CC;

typedef struct two_float {
	float x;
	float y;
} FF;

typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;

typedef struct compare_block {
	II middle;
	II up;
	II down;
	II left;
	II right;
}CB;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>hills_location(0, one_side_number);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 10);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(5, max_height);

uniform_int_distribution <int>wind_speed_uid(0, 50);
uniform_int_distribution <int>wind_angle_uid(0, 360);

__global__
void make_hills_cuda(char** terrain_array_device, HI* hill_location_device, int hill_number)
{
	II terrain;
	terrain.x = blockIdx.x * blockDim.x + threadIdx.x;
	terrain.y = blockIdx.y * blockDim.y + threadIdx.y;
	HI hill;
	int distance;
	//printf("%d %d %d\n", terrain.x, terrain.y, i);
	//terrain_array_device[terrain.x][terrain.y] = base_floor;
	for (int i = 0; i < hill_number; i++) {
		//���� ��� ä���
		hill.x = hill_location_device[i].x;
		hill.y = hill_location_device[i].y;
		hill.radius = hill_location_device[i].radius;
		hill.height = hill_location_device[i].height;
		distance = sqrt(pow(terrain.y - hill.y, 2) + pow(terrain.x - hill.x, 2));

		if (distance <= hill.radius) {
			terrain_array_device[terrain.x][terrain.y] += (hill.height) * (hill.radius - distance) / hill.radius;
			return;
		}
	}
	
	
	////int id = threadIdx.x + blockIdx.x * blockDim.x;
	//int my_hill = threadIdx.x;
	//int my_y = blockIdx.x;
	//int hill_location_x = hill_location_device[my_hill].x;
	//int hill_location_y = hill_location_device[my_hill].y;
	//int radius = hill_location_device[my_hill].radius;
	//int height = hill_location_device[my_hill].height;
	//int distance{};
	//for (int x = 0; x < one_side_number; x++) {
	//	if (x < 0 || x >= one_side_number)
	//		continue;
	//	distance = sqrt(pow(x - hill_location_x, 2) + pow(my_y - hill_location_y, 2));
	//	if (distance <= radius) {
	//		terrain_array_device[x][my_y] = (height - 1) * (radius - distance) / radius;
	//		if (terrain_array_device[x][my_y] > max_height) {
	//			terrain_array_device[x][my_y] = max_height;
	//		}
	//	}
	//}
}

__global__
void terrain_change_cuda(char** terrain_array_device, int height)
{
	//�ٶ��� �Ҹ� �������� �̵��ϰԲ� �����ؾ� ��
	//�ƴϸ� �ٶ����� ���� �Լ� ���� ���� �ߺ����� ���� �� ��� Ȯ��
	const int block_num = 9;
	II terrain[block_num];
	terrain[0].x = blockIdx.x * blockDim.x + threadIdx.x;	//middle
	terrain[0].y = blockIdx.y * blockDim.y + threadIdx.y;

	if (terrain_array_device[terrain[0].x][terrain[0].y] != height) {	//���ڷ� ���� ���̰� �ƴϸ� ����
		return;
	}
	if (terrain_array_device[terrain[0].x][terrain[0].y] <= base_floor) {	//base floor ���� ������ ���̻� ���߸� �ȵ�
		return;
	}
	
	terrain[1] = { terrain[0].x, terrain[0].y - 1 };		//up
	terrain[2] = { terrain[0].x, terrain[0].y + 1 };		//down
	terrain[3] = { terrain[0].x - 1, terrain[0].y };		//left
	terrain[4] = { terrain[0].x + 1, terrain[0].y };		//right
	terrain[5] = { terrain[0].x - 1, terrain[0].y - 1 };	//up left
	terrain[6] = { terrain[0].x - 1, terrain[0].y + 1 };	//down left
	terrain[7] = { terrain[0].x + 1, terrain[0].y - 1 };	//up right
	terrain[8] = { terrain[0].x + 1, terrain[0].y + 1 };	//down right

	int height_difference = 0;
	for (int i = 1; i < block_num; i++) {	//�ֺ� ������ ���� ���̸� ����
		if (terrain[i].x < 0 || terrain[i].x >= one_side_number || terrain[i].y < 0 || terrain[i].y >= one_side_number) {	//i��° ���� �� �ܺ��϶�
			continue;
		}
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] > height_difference) {
			height_difference = terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y];
		}
	}

	if (height_difference <= 0) {	//�ֺ��� �� ���� ���� ������ ����
		return;
	}

	int num_of_lowest{};
	for (int i = 1; i < block_num; i++) {
		if (terrain[i].x < 0 || terrain[i].x >= one_side_number || terrain[i].y < 0 || terrain[i].y >= one_side_number) {	//i��° ���� �� �ܺ��϶�
			num_of_lowest++;
			continue;
		}
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
			num_of_lowest++;
		}
	}

	if (num_of_lowest == 1) {	//���� ���� ���̰� �ϳ��� �ű�� �̵�
		for (int i = 1; i < block_num; i++) {
			if (terrain[i].x < 0 || terrain[i].x >= one_side_number || terrain[i].y < 0 || terrain[i].y >= one_side_number) {	//i��° ���� �� �ܺ��϶�
				continue;
			}
			if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
				terrain_array_device[terrain[i].x][terrain[i].y]++;
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
		}
	}
	if (num_of_lowest == 8 && height_difference == 1) {	//�ֺ��� ��� ���̰� ���� ���� ���̰� 1�̸� ����
		return;
	}
	
	char thread_seed = (terrain[0].x + terrain[0].y + abs(terrain[0].x + terrain[0].y * height)) % num_of_lowest;	//random seed made by myself
	int iter = 0;
	for (int i = 0; i < block_num; i++) {
		if (terrain[i].x < 0 || terrain[i].x >= one_side_number || terrain[i].y < 0 || terrain[i].y >= one_side_number) {	//i��° ���� �� �ܺ��϶�
			if (thread_seed == iter) {	//�������� ���õ� ������ �̵�
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
			iter++;
		}
		else if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {	//���� ���� ���̰� �������� �������� �ϳ� ����
			if (thread_seed == iter) {	//�������� ���õ� ������ �̵�
				terrain_array_device[terrain[i].x][terrain[i].y]++;
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
			iter++;
		}
	}
}

__global__
void wind_blow_cuda(char** terrain_array_device, FF wind_direction, int wind_speed)
{
	//wind_direction�� x, y�� �ϳ��� ������ 0�̿��� ��
	const int block_num = 4;
	II terrain[block_num];
	terrain[0].x = blockIdx.x * blockDim.x + threadIdx.x;	//middle
	terrain[0].y = blockIdx.y * blockDim.y + threadIdx.y;
	terrain[1] = { terrain[0].x, terrain[0].y };
	
	terrain[1].x += wind_direction.x;	//�̵������� ���� ��
	terrain[1].y += wind_direction.y;
	terrain[2] = terrain[1];	//�̵������� ���� �밢�� ��
	terrain[3] = terrain[1];	//�̵������� ������ �밢�� ��

	
	if ((int)wind_direction.x == 0) {
		terrain[2].x -= 1;
		terrain[3].x += 1;
	}
	else if ((int)wind_direction.y == 0) {
		terrain[2].y -= 1;
		terrain[3].y += 1;
	}
	//printf("%d %d,,, %d %d, %d %d, %d %d\n", terrain[0].x, terrain[0].y, terrain[2].x, terrain[2].y, terrain[1].x, terrain[1].y, terrain[3].x, terrain[3].y);

	if (terrain[1].x < 0 || terrain[1].x >= one_side_number || terrain[1].y < 0 || terrain[1].y >= one_side_number) {	//next�� �� �ܺ��϶�
		return;
	}
	if (terrain[2].x < 0 || terrain[2].x >= one_side_number || terrain[2].y < 0 || terrain[2].y >= one_side_number) {	//next�� �� �ܺ��϶�
		return;
	}
	if (terrain[3].x < 0 || terrain[3].x >= one_side_number || terrain[3].y < 0 || terrain[3].y >= one_side_number) {	//next�� �� �ܺ��϶�
		return;
	}
	if (terrain_array_device[terrain[0].x][terrain[0].y] <= base_floor) {	//base floor ���� ������ ���̻� ���߸� �ȵ�
		return;
	}

	int height_difference = -10000;
	for (int i = 1; i < block_num; i++) {
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y]  > height_difference) {
			height_difference = terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y];	//i���� 0���� ���� ����
		}
	}
	//������ ��������, ������ 1ĭ ���̸� �ö�
	int num_of_lowest{};
	for (int i = 1; i < block_num; i++) {
		if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
			num_of_lowest++;
		}
	}
	if (num_of_lowest == 1) {	//�������̰� �ϳ��� ��� �׸��� ��
		for (int i = 1; i < block_num; i++) {
			if (terrain_array_device[terrain[0].x][terrain[0].y] - terrain_array_device[terrain[i].x][terrain[i].y] == height_difference) {
				terrain_array_device[terrain[i].x][terrain[i].y]++;
				terrain_array_device[terrain[0].x][terrain[0].y]--;
				return;
			}
		}
	}
	if (num_of_lowest == 3) {	//��,����,���� ���̰� �������̷� ���� ��, ���� �켱 �̵�
		terrain_array_device[terrain[1].x][terrain[1].y]++;
		terrain_array_device[terrain[0].x][terrain[0].y]--;
	}
	//�������� ��� �������� ��
	//������ ���Ƽ� �ߺ�ó�� �ɼ������ϱ� �ٶ� ���⿡�� one side num ��ŭ for������ ������ cuda�����ؾ���
	//�ٶ����� �Ѿ�� �ſ� �켱����: ���̰� ������ ������, ���̰� �ٸ��ٸ� ����������
	
	/*if (terrain_array_device[terrain[1].x][terrain[1].y] - terrain_array_device[terrain[0].x][terrain[0].y] < 2) {
		terrain_array_device[terrain[0].x][terrain[0].y]--;
		terrain_array_device[terrain[1].x][terrain[1].y]++;
	}
	else if (terrain_array_device[terrain[2].x][terrain[2].y] - terrain_array_device[terrain[0].x][terrain[0].y] < 2) {
		terrain_array_device[terrain[0].x][terrain[0].y]--;
		terrain_array_device[terrain[2].x][terrain[2].y]++;
	}
	else if (terrain_array_device[terrain[3].x][terrain[3].y] - terrain_array_device[terrain[0].x][terrain[0].y] < 2) {
		terrain_array_device[terrain[0].x][terrain[0].y]--;
		terrain_array_device[terrain[3].x][terrain[3].y]++;
	}
	if (terrain_array_device[terrain[0].x][terrain[0].y] < 1) {
		terrain_array_device[terrain[0].x][terrain[0].y] = 1;
	}*/
}

__global__
void player_terrain_update_cuda(char** terrain_player_sight_device, HI* hill_location_device, int num_of_hills, II player_location, FF wind_direction, int wind_speed)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int terrain_x = player_location.x + x;
	int terrain_y = player_location.y + y;

	if (terrain_x >= 0 && terrain_x <= one_side_number && terrain_y >= 0 && terrain_y <= one_side_number) {
		terrain_player_sight_device[x][y] = base_floor;
		for (int i = 0; i < num_of_hills; i++) {
			
			//���� ��� ä���
			int hill_location_x = hill_location_device[i].x;
			int hill_location_y = hill_location_device[i].y;
			int radius = hill_location_device[i].radius;
			int height = hill_location_device[i].height;
			int distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

			if (distance <= radius) {
				terrain_player_sight_device[x][y] += (height) * (radius - distance) / radius;

				//��� ���
				hill_location_x = hill_location_device[i].x - radius * wind_direction.x * (100 - wind_speed) / 50;
				hill_location_y = hill_location_device[i].y - radius * wind_direction.y * (100 - wind_speed) / 50;
				distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

				if (distance <= radius) {
					terrain_player_sight_device[x][y] -= height * (radius - distance) / radius + base_floor;
					if (terrain_player_sight_device[x][y] > max_height) {
						terrain_player_sight_device[x][y] = max_height;
					}
					else if (terrain_player_sight_device[x][y] < base_floor) {
						terrain_player_sight_device[x][y] = base_floor;
					}
				}
			}
		}
	}
	else {
		//�� ���� ��� 0
		terrain_player_sight_device[x][y] = 0;
	}
}

class Map
{
private:
	char** terrain_array_host = new char* [one_side_number];
	char** terrain_array_device;
	char* terrain_array_temp[one_side_number];

	char** terrain_player_sight_host = new char* [player_sight_size];

	int num_of_hills;
	int origin_num_of_hills;

public:
	Map()
	{
		//Make Random Hills Information===================================================
		clock_t t_0 = clock();

		HI* hill_location_host;
		HI* hill_location_device;
		hill_location_host = new HI[4000];
		hill_location_device;
		hipMalloc((void**)&hill_location_device, 4000 * sizeof(HI));
		num_of_hills = make_hill_location(hill_location_host);
		origin_num_of_hills = num_of_hills;
		hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device
		printf("Random Hill Info Complete\n");
		for (int i = 0; i < num_of_hills; i++) {
			cout << hill_location_host[i].x << ", " << hill_location_host[i].y << ", " << hill_location_host[i].height << ", " << hill_location_host[i].radius << endl;
		}


		//Terrain Memory Assignement===================================================
		clock_t t_1 = clock();
		for (int i = 0; i < one_side_number; i++) {
			terrain_array_host[i] = new char[one_side_number];
		}
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				terrain_array_host[i][j] = 1;
			}
		}
		hipMalloc((void**)&terrain_array_device, one_side_number * sizeof(char*));
		for (int i = 0; i < one_side_number; i++) {
			hipMalloc((void**)&terrain_array_temp[i], one_side_number * sizeof(char));
		}
		hipMemcpy(terrain_array_device, terrain_array_temp, one_side_number * sizeof(char*), hipMemcpyHostToDevice);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_temp[i], terrain_array_host[i], one_side_number * sizeof(char), hipMemcpyHostToDevice);
		}


		//Terrain Memory Assignment For Player's Sight===================================================
		clock_t t_2 = clock();
		for (int i = 0; i < player_sight_size; i++) {
			terrain_player_sight_host[i] = new char[player_sight_size];
		}
		for (int i = 0; i < player_sight_size; i++) {
			for (int j = 0; j < player_sight_size; j++) {
				terrain_player_sight_host[i][j] = 0;
			}
		}


		//Make Hills===================================================
		clock_t t_3 = clock();
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		make_hills_cuda << <grid, block >> > (terrain_array_device, hill_location_device, num_of_hills);
		//make_hills_cuda << <one_side_number, num_of_hills >> > (terrain_array_device, hill_location_device);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		printf("Terrain Generation Complete\n");
		clock_t  t_4 = clock();

		cout << "Terrain size : " << one_side_number << " * " << one_side_number << endl;
		cout << "Terrain Array Size : " << one_side_number * one_side_number * sizeof(char) << " Bytes" << endl;
		cout << "Make Random Hills Information : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Terrain Memory Assignement : " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Terrain Memory Assignment For Player's Sight : " << (double)(t_3 - t_2) / CLOCKS_PER_SEC << " sec" << endl;
		cout << "Make Hills : " << (double)(t_4 - t_3) / CLOCKS_PER_SEC << " sec" << endl;
		cout << endl;

	}

	~Map()
	{
		for (int i = 0; i < one_side_number; i++) {
			delete[] terrain_array_host[i];
		}
		for (int i = 0; i < player_sight_size; i++) {
			delete[] terrain_player_sight_host[i];
		}
		delete[] terrain_array_host;
		delete[] terrain_player_sight_host;
		hipFree(terrain_array_temp);
		hipFree(terrain_array_device);
	}
	
	CC get_highest_lowest()
	{
		clock_t t_0 = clock();
		char highest = terrain_array_host[0][0];
		char lowest = terrain_array_host[0][0];
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				if (terrain_array_host[i][j] > highest) {
					highest = terrain_array_host[i][j];
				}
				else if (terrain_array_host[i][j] < lowest) {
					lowest = terrain_array_host[i][j];
				}
			}
		}
		CC value{ highest, lowest };
		clock_t t_1 = clock();
		cout << "Get Highest : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
		return value;
	}

	void terrain_change()
	{
		clock_t t_0 = clock();
		CC hi_low = get_highest_lowest();
		cout << hi_low.x << " " << hi_low.y << endl;
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		for (int height = hi_low.x; height > hi_low.y; height--) {
			for (int j = 0; j < 1; j++) {
				terrain_change_cuda << <grid, block >> > (terrain_array_device, height);
			}
		}
		
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		
		clock_t t_1 = clock();
		cout << "Terrain Flatten : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}

	void wind_blow(int wind_angle, int wind_speed)
	{
		clock_t t_0 = clock();
		//1m �̵��̸� �� 4ĭ�̴�, ǳ���� 90�� ������ �ֵ�, ���� ������ ������ ������Ѽ� ���ϴ� ǳ�⿡ �����.
		//���� 3�� �������� �ְ� 1�� ���� �ָ�, �̵��Ÿ��� 1m�̰�, ǳ���� 25���� �ȴ�.
		//2���� �迭�̶�, ��Ȯ���� ������ �ٻ��� ��.
		
		FF wind_direction = { cos(wind_angle * PI / 180), sin(wind_angle * PI / 180) };
		if (abs(wind_direction.x) < FLT_EPSILON) {
			wind_direction.x = 0;
		}
		if (abs(wind_direction.y) < FLT_EPSILON) {
			wind_direction.y = 0;
		}
		
		dim3 grid(one_side_number / 32, one_side_number / 32, 1);
		dim3 block(32, 32, 1);
		wind_blow_cuda << <grid, block >> > (terrain_array_device, wind_direction, wind_speed);
		for (int i = 0; i < one_side_number; i++) {
			hipMemcpy(terrain_array_host[i], terrain_array_temp[i], one_side_number * sizeof(char), hipMemcpyDeviceToHost);
		}
		clock_t t_1 = clock();
		cout << "Wind Blow : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}
	
	void add_all()
	{
		clock_t t_0 = clock();
		unsigned int all = 0;
		for (int i = 0; i < one_side_number; i++) {
			for (int j = 0; j < one_side_number; j++) {
				all += terrain_array_host[i][j];
			}
		}

		clock_t t_1 = clock();
		cout << all << endl;
		cout << "Terrain Add All : " << (double)(t_1 - t_0) / CLOCKS_PER_SEC << " sec" << endl;
	}
	
	void show_array(char** terrain_array_host, int size)
	{
		for (int y = 0; y < size; y++) {
			for (int x = 0; x < size; x++) {
				printf("%d ", terrain_array_host[x][y]);
			}
			printf("\n");
		}
	}

	void copy_for_player_map(II player_location)
	{
		clock_t start_t, end_t;
		start_t = clock();
		for (int i = 0; i < player_sight_size; i++) {
			for (int j = 0; j < player_sight_size; j++) {
				if (player_location.x - player_sight_size / 2 + i < 0 || player_location.x - player_sight_size / 2 + i >= one_side_number || player_location.y - player_sight_size / 2 + j < 0 || player_location.y - player_sight_size / 2 + j >= one_side_number)
					terrain_player_sight_host[i][j] = 0;
				else
					terrain_player_sight_host[i][j] = terrain_array_host[player_location.x - player_sight_size / 2 + i][player_location.y - player_sight_size / 2 + j];
			}
		}
		end_t = clock();
		cout << "copy_for_player_map : " << double(end_t - start_t) / CLOCKS_PER_SEC << endl;
	}
	
	void update_player_sight()
	{
		//Terrain move & Player Sight Update===================================================
		//for (int i = 0; i < 1; i++) {
		//	clock_t t_1 = clock();

		//	//Terrain Move
		//	wind_decide(wind_speed, wind_angle);

		//	FF wind_direction = { cos(wind_angle * PI / 180), sin(wind_angle * PI / 180) };
		//	if (abs(wind_direction.x) < FLT_EPSILON) {
		//		wind_direction.x = 0;
		//	}
		//	if (abs(wind_direction.y) < FLT_EPSILON) {
		//		wind_direction.y = 0;
		//	}

		//	move_terrain(hill_location_host, num_of_hills, wind_direction, wind_speed);
		//	if (num_of_hills < origin_num_of_hills) {
		//		make_new_hills(hill_location_host, num_of_hills, origin_num_of_hills, wind_direction, wind_speed);
		//	}

		//	hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device

		//	//Player Sight Update
		//	//player_location.x += 20;
		//	//player_location.y += 20;
		//	//thread must be 1024 for efficiency
		//	player_terrain_update_cuda << <player_sight_size, player_sight_size >> > (terrain_player_sight_device, hill_location_device, num_of_hills, player_location, wind_direction, wind_speed);
		//	for (int i = 0; i < player_sight_size; i++) {
		//		hipMemcpy(terrain_player_sight_host[i], terrain_player_sight_temp[i], player_sight_size * sizeof(char), hipMemcpyDeviceToHost);
		//	}
		//	clock_t t_2 = clock();
		//	cout << "Player Sight Update Time : " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " Seconds" << endl;
		//	//show_array(terrain_player_sight_host, player_sight_size);
		//	cout << "==============================" << endl;
		//}
	}
	
	void get_device_info()
	{
		hipDeviceProp_t  prop;

		int count;
		hipGetDeviceCount(&count);

		for (int i = 0; i < count; i++) {
			hipGetDeviceProperties(&prop, i);
			printf("   --- General Information for device %d ---\n", i);
			printf("Name:  %s\n", prop.name);
			printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
			printf("Clock rate:  %d\n", prop.clockRate);
			printf("Device copy overlap:  ");
			if (prop.deviceOverlap)
				printf("Enabled\n");
			else
				printf("Disabled\n");
			printf("Kernel execution timeout :  ");
			if (prop.kernelExecTimeoutEnabled)
				printf("Enabled\n");
			else
				printf("Disabled\n");
			printf("\n");

			printf("   --- Memory Information for device %d ---\n", i);
			printf("Total global mem:  %ld\n", prop.totalGlobalMem);
			printf("Total constant Mem:  %ld\n", prop.totalConstMem);
			printf("Max mem pitch:  %ld\n", prop.memPitch);
			printf("Texture Alignment:  %ld\n", prop.textureAlignment);
			printf("\n");

			printf("   --- MP Information for device %d ---\n", i);
			printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
			printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
			printf("Registers per mp:  %d\n", prop.regsPerBlock);
			printf("Threads in warp:  %d\n", prop.warpSize);
			printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
			printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
			printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
			printf("\n");
		}
	}
	
	template <typename T>
	int delete_array(T* array, int i, int size)
	{
		for (int j = i; j < size; j++) {
			array[j] = array[j + 1];
		}
		size -= 1;
		return size;
	}

	int make_hill_location(HI* hill_location_host)
	{
		num_of_hills = number_of_hills_uid(dre);
		cout << "expected num of hills: " << num_of_hills << endl;

		for (int i = 0; i < num_of_hills; i++) {
			hill_location_host[i].x = hills_location(dre);
			hill_location_host[i].y = hills_location(dre);
			hill_location_host[i].radius = hill_size_uid(dre);
			hill_location_host[i].height = height_uid(dre);
		}
		//sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
		for (int a = 0; a < num_of_hills; a++) {
			for (int b = 0; b < num_of_hills; b++) {
				if (a != b) {
					if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
							num_of_hills = delete_array(hill_location_host, b, num_of_hills);
							b--;
						}
					}
				}
			}
		}
		//for (int i = 0; i < num_of_hills; i++) {
		//	if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
		//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
		//		--i;
		//		continue;
		//	}
		//	else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
		//		num_of_hills = delete_array (hill_location_host, i, num_of_hills);
		//		--i;
		//		continue;
		//	}
		//	else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
		//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
		//		--i;
		//		continue;
		//	}
		//	else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
		//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
		//		--i;
		//		continue;
		//	}
		//}
		cout << "real num of hills: " << num_of_hills << endl;
		return num_of_hills;
	}

	void make_new_hills(HI* hill_location_host, int& num_of_hills, int origin_num_of_hills, FF wind_direction, int wind_speed)
	{
		cout << "Wind info: " << wind_direction.x << " " << wind_direction.y << endl;
		if (wind_speed) {
			hill_location_host[num_of_hills].radius = hill_size_uid(dre);
			hill_location_host[num_of_hills].height = height_uid(dre);

			hill_location_host[num_of_hills].x = hills_location(dre);
			hill_location_host[num_of_hills].y = hills_location(dre);

			while (1) {
				hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
				hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
				if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
					break;
				}
				if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
					break;
				}
				if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
					break;
				}
				if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
					break;
				}
			}

			int collide_iter{};
			for (int a = 0; a < num_of_hills; a++) {
				if (collide_iter > 10) {	//���ѷ��� ���� ���ɼ����� ���� Ƚ�� ����
					return;
				}
				//cout << "���: "<<a << " " << hill_location_host[a].x << " " << hill_location_host[a].y << " " << hill_location_host[a].radius << " " << hill_location_host[a].height << endl;
				//cout << " ��: "<< num_of_hills<< " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills ].y << " " << hill_location_host[num_of_hills ].radius << " " << hill_location_host[num_of_hills].height << endl << endl;

				if (pow(hill_location_host[a].x - hill_location_host[num_of_hills].x, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {
					if (pow(hill_location_host[a].y - hill_location_host[num_of_hills].y, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {

						hill_location_host[num_of_hills].radius = hill_size_uid(dre);
						hill_location_host[num_of_hills].height = height_uid(dre);

						hill_location_host[num_of_hills].x = hills_location(dre);
						hill_location_host[num_of_hills].y = hills_location(dre);

						while (1) {
							hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
							hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
							if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
								break;
							}
							if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
								break;
							}
							if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
								break;
							}
							if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
								break;
							}
						}
						cout << "�浹�� ���� �ٲ�" << endl;
						a = -1;
						collide_iter++;
					}
				}

			}
			cout << " ����: " << num_of_hills << " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills].y << " " << hill_location_host[num_of_hills].radius << " " << hill_location_host[num_of_hills].height << endl << endl;
			num_of_hills++;
		}
	}

	void move_terrain(HI* hill_location_host, int& num_of_hills, FF wind_direction, int wind_speed)
	{
		if (wind_speed) {
			int wind_move_x = wind_speed * wind_direction.x;
			int wind_move_y = wind_speed * wind_direction.y;
			cout << "Wind == " << "X: " << wind_move_x << " " << "Y: " << wind_move_y << endl;
			for (int i = 0; i < num_of_hills; i++) {
				hill_location_host[i].x += wind_move_x;
				hill_location_host[i].y += wind_move_y;

				if (hill_location_host[i].x - hill_location_host[i].radius > one_side_number) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].x + hill_location_host[i].radius < 0) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].y - hill_location_host[i].radius > one_side_number) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
				if (hill_location_host[i].y + hill_location_host[i].radius < 0) {
					num_of_hills = delete_array(hill_location_host, i, num_of_hills);
					--i;
					continue;
				}
			}
			/*for (int i = 0; i < num_of_hills; i++) {
				cout << hill_location_host[i].x << " " << hill_location_host[i].y << endl;
			}
			cout << endl;*/
		}
	}

	void wind_decide(int& wind_speed, int& wind_angle)
	{
		//wind speed 0-50
		//wind angle 0-360
		wind_speed = 50;// wind_speed_uid(dre);
		//wind_angle = 90; // wind_angle_uid(dre);
		wind_angle += 10;
		cout << wind_speed << " " << wind_angle << endl;
		//ǳ���� �������� �ѹ� ������Ʈ �� ������, ǳ���� �������� �ѹ� ������Ʈ �� ������ ȸ�Ǹ� ���� ��������
	}
	
	char** get_map() {
		return terrain_array_host;
	}

	char** get_player_sight_map() {
		return terrain_player_sight_host;
	}
};
