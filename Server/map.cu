#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <random>
#include <windows.h>
#include <time.h>
#include <math.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include ""
#define PI 3.1415926
using namespace std;

const int one_side_number = 40000;	//39936
const int player_sight_size = 1000;	//1024 ������ �ȵ�

const int max_height = 8;
const int base_floor = 1;

typedef struct two_int {
	int x;
	int y;
} II;

typedef struct two_float {
	float x;
	float y;
} FF;

typedef struct hill_info {
	int x;
	int y;
	int radius;
	int height;
} HI;

random_device rd;
default_random_engine dre(rd());
uniform_int_distribution <int>hills_location(0, one_side_number);
uniform_int_distribution <int>number_of_hills_uid(one_side_number / 10, one_side_number / 10);
uniform_int_distribution <int>hill_size_uid(one_side_number / 20, one_side_number / 10);
uniform_int_distribution <int>height_uid(5, max_height);

uniform_int_distribution <int>wind_speed_uid(0, 50);
uniform_int_distribution <int>wind_angle_uid(0, 360);


void get_device_info()
{
	hipDeviceProp_t  prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
		printf("\n");

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}

void show_array(char** terrain_array_host, int size)
{
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			printf("%d ", terrain_array_host[x][y]);
		}
		printf("\n");
	}
}

__global__
void player_terrain_update_cuda(char** terrain_player_sight_device, HI* hill_location_device, int num_of_hills, II player_location, FF wind_direction, int wind_speed)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int terrain_x = player_location.x + x;
	int terrain_y = player_location.y + y;

	if (terrain_x >= 0 && terrain_x <= one_side_number && terrain_y >= 0 && terrain_y <= one_side_number) {
		terrain_player_sight_device[x][y] = base_floor;
		for (int i = 0; i < num_of_hills; i++) {

			//���� ��� ä���
			int hill_location_x = hill_location_device[i].x;
			int hill_location_y = hill_location_device[i].y;
			int radius = hill_location_device[i].radius;
			int height = hill_location_device[i].height;
			int distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

			if (distance <= radius) {
				terrain_player_sight_device[x][y] += (height) * (radius - distance) / radius;

				//��� ���
				hill_location_x = hill_location_device[i].x - radius * wind_direction.x * (100 - wind_speed) / 50;
				hill_location_y = hill_location_device[i].y - radius * wind_direction.y * (100 - wind_speed) / 50;
				distance = sqrt(pow(terrain_y - hill_location_y, 2) + pow(terrain_x - hill_location_x, 2));

				if (distance <= radius) {
					terrain_player_sight_device[x][y] -= height * (radius - distance) / radius + base_floor;
					if (terrain_player_sight_device[x][y] > max_height) {
						terrain_player_sight_device[x][y] = max_height;
					}
					else if (terrain_player_sight_device[x][y] < base_floor) {
						terrain_player_sight_device[x][y] = base_floor;
					}
				}
			}
		}
	}
	else {
		//�� ���� ��� 0
		terrain_player_sight_device[x][y] = 0;
	}
}

template <typename T>
int delete_array(T* array, int i, int size)
{
	for (int j = i; j < size; j++) {
		array[j] = array[j + 1];
	}
	size -= 1;
	return size;
}

int make_hill_location(HI* hill_location_host)
{
	int num_of_hills = number_of_hills_uid(dre);
	cout << "expected num of hills: " << num_of_hills << endl;

	for (int i = 0; i < num_of_hills; i++) {
		hill_location_host[i].x = hills_location(dre);
		hill_location_host[i].y = hills_location(dre);
		hill_location_host[i].radius = hill_size_uid(dre);
		hill_location_host[i].height = height_uid(dre);
	}
	//sort(&hill_location_host[0], &hill_location_host[num_of_hills], [](const HI& a, const HI& b) { return a.y < b.y; });
	for (int a = 0; a < num_of_hills; a++) {
		for (int b = 0; b < num_of_hills; b++) {
			if (a != b) {
				if (pow(hill_location_host[a].x - hill_location_host[b].x, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
					if (pow(hill_location_host[a].y - hill_location_host[b].y, 2) - pow(hill_location_host[a].radius + hill_location_host[b].radius, 2) < 0) {
						num_of_hills = delete_array(hill_location_host, b, num_of_hills);
						b--;
					}
				}
			}
		}
	}
	//for (int i = 0; i < num_of_hills; i++) {
	//	if (hill_location_host[i].x - hill_location_host[i].radius < 0) {
	//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//	else if (hill_location_host[i].x + hill_location_host[i].radius >= one_side_number) {
	//		num_of_hills = delete_array (hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//	else if (hill_location_host[i].y - hill_location_host[i].radius < 0) {
	//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//	else if (hill_location_host[i].y + hill_location_host[i].radius >= one_side_number) {
	//		num_of_hills = delete_array(hill_location_host, i, num_of_hills);
	//		--i;
	//		continue;
	//	}
	//}
	cout << "real num of hills: " << num_of_hills << endl;
	return num_of_hills;
}

void make_new_hills(HI* hill_location_host, int& num_of_hills, int origin_num_of_hills, FF wind_direction, int wind_speed)
{
	cout << "Wind info: " << wind_direction.x << " " << wind_direction.y << endl;
	if (wind_speed) {
		hill_location_host[num_of_hills].radius = hill_size_uid(dre);
		hill_location_host[num_of_hills].height = height_uid(dre);

		hill_location_host[num_of_hills].x = hills_location(dre);
		hill_location_host[num_of_hills].y = hills_location(dre);

		while (1) {
			hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
			hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
			if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
				break;
			}
			if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
				break;
			}
			if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
				break;
			}
			if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
				break;
			}
		}

		int collide_iter{};
		for (int a = 0; a < num_of_hills; a++) {
			if (collide_iter > 10) {	//���ѷ��� ���� ���ɼ����� ���� Ƚ�� ����
				return;
			}
			//cout << "���: "<<a << " " << hill_location_host[a].x << " " << hill_location_host[a].y << " " << hill_location_host[a].radius << " " << hill_location_host[a].height << endl;
			//cout << " ��: "<< num_of_hills<< " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills ].y << " " << hill_location_host[num_of_hills ].radius << " " << hill_location_host[num_of_hills].height << endl << endl;

			if (pow(hill_location_host[a].x - hill_location_host[num_of_hills].x, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {
				if (pow(hill_location_host[a].y - hill_location_host[num_of_hills].y, 2) - pow(hill_location_host[a].radius + hill_location_host[num_of_hills].radius, 2) <= 0) {

					hill_location_host[num_of_hills].radius = hill_size_uid(dre);
					hill_location_host[num_of_hills].height = height_uid(dre);

					hill_location_host[num_of_hills].x = hills_location(dre);
					hill_location_host[num_of_hills].y = hills_location(dre);

					while (1) {
						hill_location_host[num_of_hills].x -= wind_direction.x * wind_speed;
						hill_location_host[num_of_hills].y -= wind_direction.y * wind_speed;
						if (hill_location_host[num_of_hills].x - hill_location_host[num_of_hills].radius > one_side_number) {
							break;
						}
						if (hill_location_host[num_of_hills].x + hill_location_host[num_of_hills].radius < 0) {
							break;
						}
						if (hill_location_host[num_of_hills].y - hill_location_host[num_of_hills].radius > one_side_number) {
							break;
						}
						if (hill_location_host[num_of_hills].y + hill_location_host[num_of_hills].radius < 0) {
							break;
						}
					}
					cout << "�浹�� ���� �ٲ�" << endl;
					a = -1;
					collide_iter++;
				}
			}

		}
		cout << " ����: " << num_of_hills << " " << hill_location_host[num_of_hills].x << " " << hill_location_host[num_of_hills].y << " " << hill_location_host[num_of_hills].radius << " " << hill_location_host[num_of_hills].height << endl << endl;
		num_of_hills++;
	}
}

void move_terrain(HI* hill_location_host, int& num_of_hills, FF wind_direction, int wind_speed)
{
	if (wind_speed) {
		int wind_move_x = wind_speed * wind_direction.x;
		int wind_move_y = wind_speed * wind_direction.y;
		cout << "Wind == " << "X: " << wind_move_x << " " << "Y: " << wind_move_y << endl;
		for (int i = 0; i < num_of_hills; i++) {
			hill_location_host[i].x += wind_move_x;
			hill_location_host[i].y += wind_move_y;

			if (hill_location_host[i].x - hill_location_host[i].radius > one_side_number) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			if (hill_location_host[i].x + hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			if (hill_location_host[i].y - hill_location_host[i].radius > one_side_number) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
			if (hill_location_host[i].y + hill_location_host[i].radius < 0) {
				num_of_hills = delete_array(hill_location_host, i, num_of_hills);
				--i;
				continue;
			}
		}
		/*for (int i = 0; i < num_of_hills; i++) {
			cout << hill_location_host[i].x << " " << hill_location_host[i].y << endl;
		}
		cout << endl;*/
	}
}

void wind_decide(int& wind_speed, int& wind_angle)
{
	//wind speed 0-50
	//wind angle 0-360
	wind_speed = 50;// wind_speed_uid(dre);
	//wind_angle = 90; // wind_angle_uid(dre);
	wind_angle += 10;
	cout << wind_speed << " " << wind_angle << endl;
	//ǳ���� �������� �ѹ� ������Ʈ �� ������, ǳ���� �������� �ѹ� ������Ʈ �� ������ ȸ�Ǹ� ���� ��������


}
//
//int main()
//{
//	//get_device_info();
//
//
//	//ToDo
//	//���� ��ġ ���� ����, ���� ��ġ�� ���� 0�ε� �ֺ� ����� �������� ���̷���, ��� �����̱�
//	// ����� �����̴µ� ȿ������ ���: �÷��̾ ���� �þ߸� ������Ʈ�� �ϰԲ� �ؾ��� 
//	// hill_location ������Ʈ�� �׻� �ϰ�, cuda�� �÷��̾ ���� ������ terrain_array�� ������Ʈ �ؾ���
//	// �÷��̾ ���� ������ 1000*1000�̶� �����ϰ�, �� �κ��� terrain_array�� ������Ʈ �ؾ���
//	// �÷��̾� ���� �°� �÷��̾� ���� 2���� �迭 terrain_array_for_player�� �����Ҵ��Ͽ� �����ؾ���
//	// 
//	// �÷��̾�� �� ������ ������� �� �ñ�: ���� ����, ������ ����� ��, ī�޶� ������ ��
//	// ����: ������ �����϶� terrain_array��ü�� �����ϰ�, �÷��̾�� terrain_array�� ���縸 �ؼ� �Ϻκ��� ��������,
//	// hill_location�� �����ϰ�, �÷��̾�� �����ٶ�, terrain_array�� �ƿ� �׸���. ====�̰� ������...40000*40000 ����°� �����̿���..
//	//
//	// ������ �����̸�, �����̴� ������ �ݴ� �κ��� ��� �ȴ�. ����� ���ٴ��� ���...
//	// ������ �� �������� �����̸� �����������
//	// �׸��� 40000 * 40000 ������ ���� �����  hill_location �迭���� ���������.
//	// 
//	// ����� �����ɶ� ����� ���ĸ� ������ �����غ��� �͵� ������ �ϴ�
//	// 
//	// �縷�� �𷡾�� ����� ������ ���� �ȴٸ� ������~�ʽ´��� ����� ��� �Ǵ°Ͱ���
//	// �ٶ��� ����� ���⿡ ���� �� ����� ���ϴ� ������ �ٲ�����Ѵ�.
//	// �ٶ��� �ѹ��� ���� �ʾҴٸ� �� ����� ��ٴ� �����Ͽ� �ٶ��� �Ҹ�, �ٶ��� �� �������� ����ִ� ����� ������
//	// �ٶ��� ���ʿ��� �дٸ� ����� ���� �κ��� ��� ����� ���������� �̵��ϰԲ� �Ѵ�,
//	// �� �� �ٶ��� ���ʿ��� �дٸ� ��ĭ�� ������ ���Եǰ�, �������� ���̴� �������� ������.
//	// �ٶ��� ������ ��ġ�� ����: ����� �𿴴� �κ��� ä������ ����, ����� ���̴� ����
//	// �ٶ��� ���Ⱑ ��ġ�� ����: ������ �̵��ϴ� �ӵ�, ����� ���̴� �ӵ�
//	// 
//	// //���� ����� ���̴� ����� ����� ���̰� �Ϸ��� ǳ�Ӱ� ǳ���� ȮȮ �ٲٸ� �ȵȴ�.
//	// �ִ� ǳ���� 50�̸� �ٶ��� �Ⱥδ� ���¿��� 10, 20, 30, 40, 50 �̷��� ���� �÷��� Ŭ������ ������� �Ѵ�
//	// �ƴϸ� ����� ����� �ѹ��� Ȯ ���Ѵ�
//	// 
//	// ���� ���� ���� ���� ������ 1:1:0.2�� �ϸ� ��� ������ ��.
//	// ���� 1:1:1�̶�� �پ��ִ� ���� ���� 0, 45�� �ۿ� ����
//	// 
//	// ������ �ٶ��� ���� �̵��ϸ�, �̵��� ������ �ݴ� ���⿡ ����� ���� �����Ǿ�� �Ѵ�.
//	// ó���� ������ ����� ������ ����ϰ�, ����� ������� �Ǹ� �׸�ŭ ���� �����ǰԲ� ����.
//	// �������� �ٶ��� �Ҿ� �����̰� �ִٸ� �����ʿ��� ����, �Ʒ��� �ٶ��� �дٸ� ������ ���� �ϰԲ�.
//	// ����� �پ �������� �ʰԲ� �ؾ��Ѵ�. �߰��� �Ҷ� hill_location �迭�� Ȯ���ؼ� �浹�˻縦 ����.
//	// 
//	// �ٶ��� �δ� �˰����� ��������, ǳ�Ӱ� ǳ���� ���������� �ٲ����Ѵ�.
//	// 
//	// 
//	// 
//	//
//
//
//	//Make Random Hills Information===================================================
//	HI* hill_location_host = new HI[4000];
//	HI* hill_location_device;
//	hipMalloc((void**)&hill_location_device, 4000 * sizeof(HI));
//	int num_of_hills = make_hill_location(hill_location_host);
//	int origin_num_of_hills = num_of_hills;
//	hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device
//	printf("Random Hill Info Complete\n");
//
//
//	//Terrain Memory Assignment For Player's Sight===================================================
//	char** terrain_player_sight_host = new char* [player_sight_size];	// 2D array for host
//	for (int i = 0; i < player_sight_size; i++) {
//		terrain_player_sight_host[i] = new char[player_sight_size];
//	}
//	for (int i = 0; i < player_sight_size; i++) {
//		for (int j = 0; j < player_sight_size; j++) {
//			terrain_player_sight_host[i][j] = 0;
//		}
//	}
//	char** terrain_player_sight_device;						// 2D array for device
//	char* terrain_player_sight_temp[player_sight_size];		// 1D array temp
//	hipMalloc((void**)&terrain_player_sight_device, player_sight_size * sizeof(char*));
//	for (int i = 0; i < player_sight_size; i++) {
//		hipMalloc((void**)&terrain_player_sight_temp[i], player_sight_size * sizeof(char));
//	}
//	hipMemcpy(terrain_player_sight_device, terrain_player_sight_temp, player_sight_size * sizeof(char*), hipMemcpyHostToDevice);
//	for (int i = 0; i < player_sight_size; i++) {
//		hipMemcpy(terrain_player_sight_temp[i], terrain_player_sight_host[i], player_sight_size * sizeof(char), hipMemcpyHostToDevice);
//	}
//
//
//	//Terrain move & Player Sight Update===================================================
//	TI player_location = { 0, 0 };		//�̰� ���߿� �߽� �������� �ٲ����
//	int wind_angle = 270;		//����
//	int wind_speed = 50;		//�ִ� ǳ�� 50
//	for (int i = 0; i < 10; i++) {
//		clock_t t_1 = clock();
//
//		//Terrain Move
//		wind_decide(wind_speed, wind_angle);
//
//		TF wind_direction = { cos(wind_angle * PI / 180), sin(wind_angle * PI / 180) };
//		if (abs(wind_direction.x) < FLT_EPSILON) {
//			wind_direction.x = 0;
//		}
//		if (abs(wind_direction.y) < FLT_EPSILON) {
//			wind_direction.y = 0;
//		}
//
//		move_terrain(hill_location_host, num_of_hills, wind_direction, wind_speed);
//		if (num_of_hills < origin_num_of_hills) {
//			make_new_hills(hill_location_host, num_of_hills, origin_num_of_hills, wind_direction, wind_speed);
//		}
//
//		hipMemcpy(hill_location_device, hill_location_host, num_of_hills * sizeof(HI), hipMemcpyHostToDevice); //Memcpy to Device
//
//		//Player Sight Update
//		//player_location.x += 20;
//		//player_location.y += 20;
//		//thread must be 1024 for efficiency
//		player_terrain_update_cuda << <player_sight_size, player_sight_size >> > (terrain_player_sight_device, hill_location_device, num_of_hills, player_location, wind_direction, wind_speed);
//		for (int i = 0; i < player_sight_size; i++) {
//			hipMemcpy(terrain_player_sight_host[i], terrain_player_sight_temp[i], player_sight_size * sizeof(char), hipMemcpyDeviceToHost);
//		}
//		clock_t t_2 = clock();
//		cout << "Player Sight Update Time : " << (double)(t_2 - t_1) / CLOCKS_PER_SEC << " Seconds" << endl;
//		//show_array(terrain_player_sight_host, player_sight_size);
//		cout << "==============================" << endl;
//	}
//
//
//
//	//File Save===================================================
//	/*ofstream out{"terrain.txt"};
//
//	for (int i = 0; i < one_side_number; i++) {
//		for (int j = 0; j < one_side_number; j++) {
//			out << terrain_array_host[i][j] << " ";
//		}
//		out << endl;
//	}
//	cout << "File Write Complete" << endl;*/
//
//
//	//Free Memory===================================================
//	delete[] hill_location_host;
//	hipFree(hill_location_device);
//	for (int i = 0; i < player_sight_size; i++) {
//		delete[] terrain_player_sight_host[i];
//		hipFree(terrain_player_sight_temp[i]);
//	}
//	delete[] terrain_player_sight_host;
//	hipFree(terrain_player_sight_device);
//	hipFree(terrain_player_sight_temp);
//}
